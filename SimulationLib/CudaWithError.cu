#include "CudaWithError.cuh"
#include<iostream>


CudaWithError::CudaWithError(string className): className(className) { }

void CudaWithError::throwErrorMaybe(hipError_t cudaStatus, string error) {
    if (cudaStatus != hipSuccess) {
        string totalError = className + ": " + error + "\n" + hipGetErrorString(cudaStatus);
        cerr << endl << totalError;
        throw new runtime_error(totalError);
    }
}

void CudaWithError::setDevice(int device) {
	hipError_t cudaStatus = hipSetDevice(device);
    throwErrorMaybe(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
}

void CudaWithError::malloc(void** devPtr, size_t size) {
    hipError_t cudaStatus = hipMalloc(devPtr, size);
    throwErrorMaybe(cudaStatus, "hipMalloc failed!");
}

void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {
    hipError_t cudaStatus = hipMemcpy(dst, src, count, kind);
    throwErrorMaybe(cudaStatus, "hipMemcpy failed!");
}

void CudaWithError::deviceSynchronize() {
    hipError_t cudaStatus = hipDeviceSynchronize();
    throwErrorMaybe(cudaStatus, "hipDeviceSynchronize failed!");
}

void CudaWithError::free(void* devPtr) {
    hipError_t cudaStatus = hipFree(devPtr);
    throwErrorMaybe(cudaStatus, "hipFree failed!");
}
