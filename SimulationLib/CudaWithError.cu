#include "CudaWithError.cuh"
#include<iostream>


CudaWithError::CudaWithError(string className): className(className) { }

void CudaWithError::throwErrorMaybe(hipError_t cudaStatus, string error) {
    if (cudaStatus != hipSuccess) {
        string totalError = className + ": " + error;
        cerr << endl << totalError;
        throw new runtime_error(totalError);
    }
}

void CudaWithError::setDevice(int device) {
	hipError_t cudaStatus = hipSetDevice(device);
    throwErrorMaybe(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
}

void CudaWithError::malloc(void** devPtr, size_t size) {

}

void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {

}

void CudaWithError::deviceSynchronize() {

}

void CudaWithError::hipFree(void* devPtr) {
    
}
