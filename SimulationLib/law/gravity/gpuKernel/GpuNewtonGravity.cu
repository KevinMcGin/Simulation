#include "hip/hip_runtime.h"
#include "law/gravity/gpuKernel/GpuNewtonGravity.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "law/gravity/helper/GpuNewtonGravityHelper.cuh"

#include <cmath>

GpuNewtonGravity::GpuNewtonGravity(double G) : GpuLaw("GpuNewtonGravity"), G(G) { }

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* accelerations, int n, double G) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		radiusComponentKernelHelper(idx, particles, accelerations, n, G);
	} 
}

__global__ 
void addAccelerationsKernelLower(Particle** particles, Vector3D* accelerations, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelLowerHelper(idx, particles, accelerations, x0, y, n);
}

__global__ 
void addAccelerationsKernelUpper(Particle** particles, Vector3D* accelerations, int x0, int y, int n, int betweenParticlesTriangularCount) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelUpperHelper(idx, particles, accelerations, x0, y, n, betweenParticlesTriangularCount);
}

void GpuNewtonGravity::run(Particle** td_par, int particleCount) {
	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount;
	int betweenParticlesTriangularCount = betweenParticlesCount/2;
	Vector3D* accelerations = NULL;
	cudaWithError->malloc((void**)&accelerations, betweenParticlesCount*sizeof(Vector3D));
	radiusComponentKernel <<<1 + betweenParticlesTriangularCount/256, 256>>> (td_par, accelerations, betweenParticlesTriangularCount, G);
	cudaWithError->peekAtLastError("radiusComponentKernel");

	for(int i = 0; i < particleCount; i++) {
		addAccelerationsKernelLower <<<1 + i/256, 256>>> (td_par, accelerations, 0, i, i);
		cudaWithError->peekAtLastError("addAccelerationsKernelLower");
		addAccelerationsKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, accelerations, i+1, i, particleCount, betweenParticlesTriangularCount);
		cudaWithError->peekAtLastError("addAccelerationsKernelUpper");
	}
	
	cudaWithError->free(accelerations);
}

