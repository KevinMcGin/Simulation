#include "hip/hip_runtime.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"

__device__ __host__
bool CollisionDetectorSimple::isCollision(Particle* p1, Particle* p2)
{
	Vector3D difference = p1->position - p2->position;
	return difference.magnitudeSquared() < pow(p1->radius + p2->radius, 2);
}
