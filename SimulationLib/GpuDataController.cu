#include "GpuDataController.cuh"
#include "CudaWithError.cuh"
#include "ParticleSimple.h"

GpuDataController::GpuDataController() {
    cudaWithError = new CudaWithError("GpuDataController");
    cudaWithError->setDevice(0);
}

GpuDataController::~GpuDataController() {
    delete cudaWithError;
}

void GpuDataController::putParticlesOnDevice(vector<Particle*> particles) {
	particleCount = (int)particles.size();
    
    //Instantiate object on the CPU
	auto particlesArray = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i)
		particlesArray[i] = particles[i];

	//Copy dynamically allocated child objects to GPU
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->malloc((void**)&d_par[i], sizeof(ParticleSimple));
		cudaWithError->memcpy(d_par[i], particlesArray[i], sizeof(ParticleSimple), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	cudaWithError->malloc((void**)&td_par, particleCount * sizeof(Particle *));
	cudaWithError->memcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);

    delete particlesArray;
}

void GpuDataController::getParticlesFromDevice(vector<Particle*>& particles) {
	int particleCount = (int)particles.size();

	//copy particles back to cpu
	for(int i = 0; i < particleCount; i++) {
        //TODO Is using the vector particles here ok?
		cudaWithError->memcpy(particles[i] ,d_par[i], sizeof(ParticleSimple), hipMemcpyDeviceToHost);
		cudaWithError->free(d_par[i]);
	}
	cudaWithError->free(td_par);
	delete d_par;
}

