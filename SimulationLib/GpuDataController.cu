#include "GpuDataController.cuh"
#include "CudaWithError.cuh"
#include "ParticleSimple.h"

GpuDataController::GpuDataController() {
    cudaWithError = new CudaWithError("GpuDataController");
    cudaWithError->setDevice(0);
}

GpuDataController::~GpuDataController() {
    delete cudaWithError;
}

void GpuDataController::putParticlesOnDevice(vector<Particle*> particles, bool firstRun) {
	if(!firstRun) {
		for(int i = 0; i < particleCount; i++) {
			cudaWithError->free(d_par[i]);
		}
		cudaWithError->free(td_par);
		delete d_par;
	}
	
	particleCount = (int)particles.size();

	//Copy dynamically allocated child objects to GPU
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->malloc((void**)&d_par[i], sizeof(ParticleSimple));
		cudaWithError->memcpy(d_par[i], particles[i], sizeof(ParticleSimple), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	cudaWithError->malloc((void**)&td_par, particleCount * sizeof(Particle *));
	cudaWithError->memcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);
}

void GpuDataController::getParticlesFromDevice(vector<Particle*>& particles) {
	for(int i = 0; i < particleCount; i++) {
		cudaWithError->memcpy(particles[i], d_par[i], sizeof(ParticleSimple), hipMemcpyDeviceToHost);
	}
}

