#include "hip/hip_runtime.h"
#include "Collision.cuh"
#include "ParticleSimple.h"
#include "CollisionDetectorSimple.cuh"
#include "CollisionResolverCoalesce.cuh"
#include "MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <set>
#include <cmath>
#include <typeinfo>


//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
__global__ 
void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
			collisionDetectorGpu[0] = new CollisionDetectorSimple();
		} else {
			printf("collisionDetectorGpu could not be initialised\n");
			assert(false);
		}
	} 
}
__global__ 
void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
			collisionResolverGpu[0] = new CollisionResolverCoalesce();
		} else {
			printf("collisionResolverGpu could not be initialised\n");
			assert(false);
		}
	} 
}

Collision::Collision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver, bool use_gpu): Law("Collision"),
	collisionDetector(collisionDetector),
	collisionResolver(collisionResolver),
	use_gpu(use_gpu) {
	if(use_gpu) {
		cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
		cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
		setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
		setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
		cudaWithError->peekAtLastError("setCollisionDetector");
	}
}

Collision::~Collision() {
	if(use_gpu) {
		cudaWithError->free(collisionDetectorGpu);
		cudaWithError->free(collisionResolverGpu);
	}
}

__global__ 
void getCollidedParticles(Particle** particles, bool* collisionMarks, CollisionDetector** collisionDetectorGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		auto p1 = particles[x];
		auto p2 = particles[y];
		if (collisionDetectorGpu[0]->isCollision(p1, p2)) {
			collisionMarks[idx] = true;
		} else {
			collisionMarks[idx] = false;			
		}
	} 
}

enum MergeStatus { LOWER_COLLISION_FOUND, COLLISION_FOUND, NO_COLLISION_FOUND };

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun = false);
__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n);

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun) {
	bool collisionsToResolve = false;
	MergeStatus mergeStatus = mergeCollisionsColumns(collisionMarks, idx, row, n);
	switch(mergeStatus) {
		case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
		case COLLISION_FOUND: collisionsToResolve = true; break;
		case NO_COLLISION_FOUND: break;
	}
	for(int i = 0; i < row; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, row);
		if(collisionMarks[collisionMarksIndex]) {
			int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
			if(firstRun || !collisionMarks[correspondingCollisionMarksIndex]) {
				collisionMarks[correspondingCollisionMarksIndex] = true;
				collisionsToResolve = true;
				mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
				switch(mergeStatus) {
					case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n) {
	bool collisionsToResolve = false;
	for(int i = row + 1; i < n; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(row, i);
		if(collisionMarks[collisionMarksIndex]) {
			if(i > idx) {
				return LOWER_COLLISION_FOUND;
			} else if(i < idx) {
				int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
				if(!collisionMarks[correspondingCollisionMarksIndex]) {
					collisionMarks[correspondingCollisionMarksIndex] = true;
					collisionsToResolve = true;
					MergeStatus mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
					switch(mergeStatus) {
						case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
					}
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__global__ 
void resolveCollidedParticles(Particle** particles, bool* collisionMarks, CollisionResolver** collisionResolverGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		auto collisionsToResolve = mergeCollisionsRows(collisionMarks, idx, idx, n, true) == COLLISION_FOUND; 
		if(collisionsToResolve) {
			auto p1 = particles[idx];
			for(int i = 0; i < idx; i++) {
				int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
				if (collisionMarks[collisionMarksIndex]) {
					auto p2 = particles[i];
					collisionResolverGpu[0]->resolve(p1, p2);
				}
			}
		}
		// delete collisionResolver;
	} 
}

void Collision::cpuRun(vector<Particle*>& particles) {
	// get particles that collided
	vector<set<Particle*>*> particlesCollidedVector;
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
    	set<Particle*> particlesCollidedSet = {};
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			if (collisionDetector->isCollision(p1, p2)) {
				particlesCollidedSet.insert(p1);
				particlesCollidedSet.insert(p2);
			}
		}
		if(particlesCollidedSet.size() > 0)
			particlesCollidedVector.push_back(new set<Particle*>(particlesCollidedSet));
	}
	// merge sets of particles that collided
	for (auto it1 = particlesCollidedVector.begin(); it1 != particlesCollidedVector.end(); it1++) {
		auto particlesCollided1 = *it1;
		for (auto it2 = it1+1; it2 < particlesCollidedVector.end(); it2++) {
			auto particlesCollided2 = *it2;
			if ([&]() {
				for(auto p: *particlesCollided2) {
					if(particlesCollided1->find(p) != particlesCollided1->end()) {
						return true;
					}
				}
				return false;
			}()) {
				for(auto p: *particlesCollided2) {
					particlesCollided1->insert(p);
				}
				particlesCollided2->clear();
			}
		}
	}
	//resolve particles
	for (auto particlesCollided1: particlesCollidedVector) {
		if(particlesCollided1->size() > 0) {
			auto p1 = *(particlesCollided1->begin());
			particlesCollided1->erase(particlesCollided1->begin());
			for(auto p2: *particlesCollided1) {
				collisionResolver->resolve(p1, p2);
			}
		}
	}
	//erase particles marked for deletion safely
	for (auto it = particles.begin(); it != particles.end();) {
		if((*it)->deleted) {
			delete *it;
			it = particles.erase(it);
		}
		else
			++it;
	}
}

void Collision::gpuRun(Particle** td_par, int particleCount) {
	// get particles that collided
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	bool* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, betweenParticlesCount*sizeof(bool));
	getCollidedParticles <<<1 + betweenParticlesCount/256, 256>>> (td_par, collisionMarks, collisionDetectorGpu, betweenParticlesCount);
	cudaWithError->peekAtLastError("getCollidedParticles");

	// merge sets of particles that collided and resolve
	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (td_par, collisionMarks, collisionResolverGpu, particleCount);
	cudaWithError->peekAtLastError("resolveCollidedParticles");

	cudaWithError->free(collisionMarks);
}