#include "hip/hip_runtime.h"
#include "Collision.cuh"
#include "ParticleSimple.h"
#include "CollisionDetectorSimple.cuh"
#include "CollisionResolverCoalesce.cuh"
#include "MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <set>
#include <cmath>
#include <typeinfo>

Collision::Collision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver): Law("Collision"),
	collisionDetector(collisionDetector),
	collisionResolver(collisionResolver) {
		
}

__global__ 
void getCollidedParticles(Particle** particles, bool* collisionMarks, int n, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		//TODO find soluation to this, creating new class with a pointer is slow on GPU threads
		//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
		// CollisionDetector* collisionDetector;
		// if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
		// 	collisionDetector = new CollisionDetectorSimple();
		// } else {
		// 	//printf("CollisionDetector could not be initialised\n");
		// 	assert(false);
		// }
		CollisionDetectorSimple collisionDetector = CollisionDetectorSimple();
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		auto p1 = particles[x];
		auto p2 = particles[y];
		if (collisionDetector.isCollision(p1, p2)) {
			collisionMarks[idx] = true;
		} else {
			collisionMarks[idx] = false;			
		}
		// delete collisionDetector;
	} 
}

enum MergeStatus { LOWER_COLLISION_FOUND, COLLISION_FOUND, NO_COLLISION_FOUND };

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun = false);
__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n);

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun) {
	bool collisionsToResolve = false;
	MergeStatus mergeStatus = mergeCollisionsColumns(collisionMarks, idx, row, n);
	switch(mergeStatus) {
		case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
		case COLLISION_FOUND: collisionsToResolve = true; break;
		case NO_COLLISION_FOUND: break;
	}
	for(int i = 0; i < row; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, row);
		if(collisionMarks[collisionMarksIndex]) {
			int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
			if(firstRun || !collisionMarks[correspondingCollisionMarksIndex]) {
				collisionMarks[correspondingCollisionMarksIndex] = true;
				collisionsToResolve = true;
				mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
				switch(mergeStatus) {
					case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n) {
	bool collisionsToResolve = false;
	for(int i = row + 1; i < n; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(row, i);
		if(collisionMarks[collisionMarksIndex]) {
			if(i > idx) {
				return LOWER_COLLISION_FOUND;
			} else if(i < idx) {
				int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
				if(!collisionMarks[correspondingCollisionMarksIndex]) {
					collisionMarks[correspondingCollisionMarksIndex] = true;
					collisionsToResolve = true;
					MergeStatus mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
					switch(mergeStatus) {
						case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
					}
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__global__ 
void resolveCollidedParticles(Particle** particles, bool* collisionMarks, int n, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		//TODO find soluation to this, creating new class with a pointer is slow on GPU threads
		//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
		// CollisionResolver* collisionResolver;
		// if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
		// 	collisionResolver = new CollisionResolverCoalesce();
		// } else {
		// 	//printf("CollisionResolver could not be initialised\n");
		// 	assert(false);
		// }
		CollisionResolverCoalesce collisionResolver = CollisionResolverCoalesce();
		auto collisionsToResolve = mergeCollisionsRows(collisionMarks, idx, idx, n, true) == COLLISION_FOUND; 
		if(collisionsToResolve) {
			auto p1 = particles[idx];
			for(int i = 0; i < idx; i++) {
				int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
				if (collisionMarks[collisionMarksIndex]) {
					auto p2 = particles[i];
					collisionResolver.resolve(p1, p2);
				}
			}
		}
		// delete collisionResolver;
	} 
}

void Collision::cpuRun(vector<Particle*>& particles) {
	// get particles that collided
	vector<set<Particle*>*> particlesCollidedVector;
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
    	set<Particle*> particlesCollidedSet = {};
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			if (collisionDetector->isCollision(p1, p2)) {
				particlesCollidedSet.insert(p1);
				particlesCollidedSet.insert(p2);
			}
		}
		if(particlesCollidedSet.size() > 0)
			particlesCollidedVector.push_back(new set<Particle*>(particlesCollidedSet));
	}
	// merge sets of particles that collided
	for (auto it1 = particlesCollidedVector.begin(); it1 != particlesCollidedVector.end(); it1++) {
		auto particlesCollided1 = *it1;
		for (auto it2 = it1+1; it2 < particlesCollidedVector.end(); it2++) {
			auto particlesCollided2 = *it2;
			if ([&]() {
				for(auto p: *particlesCollided2) {
					if(particlesCollided1->find(p) != particlesCollided1->end()) {
						return true;
					}
				}
				return false;
			}()) {
				for(auto p: *particlesCollided2) {
					particlesCollided1->insert(p);
				}
				particlesCollided2->clear();
			}
		}
	}
	//resolve particles
	for (auto particlesCollided1: particlesCollidedVector) {
		if(particlesCollided1->size() > 0) {
			auto p1 = *(particlesCollided1->begin());
			particlesCollided1->erase(particlesCollided1->begin());
			for(auto p2: *particlesCollided1) {
				collisionResolver->resolve(p1, p2);
			}
		}
	}
	//erase particles marked for deletion safely
	for (auto it = particles.begin(); it != particles.end();) {
		if((*it)->deleted) {
			delete *it;
			it = particles.erase(it);
		}
		else
			++it;
	}
}


void Collision::gpuRun(Particle** td_par, int particleCount) {
	// get particles that collided
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	bool* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, betweenParticlesCount*sizeof(bool));
	getCollidedParticles <<<1 + betweenParticlesCount/256, 256>>> (td_par, collisionMarks, betweenParticlesCount, collisionDetector->getIndex());
	cudaWithError->peekAtLastError("getCollidedParticles");

	// merge sets of particles that collided and resolve
	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (td_par, collisionMarks, particleCount, collisionResolver->getIndex());
	cudaWithError->peekAtLastError("resolveCollidedParticles");

	cudaWithError->free(collisionMarks);
}