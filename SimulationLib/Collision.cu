#include "hip/hip_runtime.h"
#include "Collision.cuh"
#include "ParticleSimple.h"
#include "CollisionDetectorSimple.cuh"
#include "CollisionResolverCoalesce.cuh"
#include "MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <set>
#include <cmath>
#include <typeinfo>

Collision::Collision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver): Law("Collision"),
	collisionDetector(collisionDetector),
	collisionResolver(collisionResolver) {
		
}

__global__ 
void getCollidedParticles(Particle** particles, bool* collisionMarks, int n, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		//printf("getCollidedParticles\n");
		//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
		// CollisionDetector* collisionDetector;
		// if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
		// 	collisionDetector = new CollisionDetectorSimple();
		// } else {
		// 	//printf("CollisionDetector could not be initialised\n");
		// 	assert(false);
		// }
		CollisionDetectorSimple collisionDetector = CollisionDetectorSimple();
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		auto p1 = particles[x];
		auto p2 = particles[y];
		//printf("Got particles\n");
		if (collisionDetector.isCollision(p1, p2)) {
			//printf("Collision. %d: %d, %d\n", idx, x, y);
			collisionMarks[idx] = true;
		} else {
			//printf("No collision\n");
			collisionMarks[idx] = false;			
		}
		// delete collisionDetector;
	} 
}

enum MergeStatus { LOWER_COLLISION_FOUND, COLLISION_FOUND, NO_COLLISION_FOUND };

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun = false);
__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n);

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun) {
	bool collisionsToResolve = false;
	MergeStatus mergeStatus = mergeCollisionsColumns(collisionMarks, idx, row, n);
	switch(mergeStatus) {
		case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
		case COLLISION_FOUND: collisionsToResolve = true; break;
		case NO_COLLISION_FOUND: break;
	}
	for(int i = 0; i < row; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndx(i, row);
		if(collisionMarks[collisionMarksIndex]) {
			int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndx(i, idx);
			if(firstRun || !collisionMarks[correspondingCollisionMarksIndex]) {
				//printf("ROWS for %d. %d: %d, %d\n", idx, collisionMarksIndex, row, i);
				collisionMarks[correspondingCollisionMarksIndex] = true;
				collisionsToResolve = true;
				mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
				switch(mergeStatus) {
					case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n) {
	bool collisionsToResolve = false;
	for(int i = row + 1; i < n; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndx(row, i);
		if(collisionMarks[collisionMarksIndex]) {
			if(i > idx) {
				//printf("LOWER_COLLISION_FOUND for %d. %d: %d, %d\n", idx, collisionMarksIndex, row, i);
				return LOWER_COLLISION_FOUND;
			} else if(i < idx) {
				int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndx(i, idx);
				if(!collisionMarks[correspondingCollisionMarksIndex]) {
					//printf("COLS for %d, %d. %d, %d: %d, %d\n", idx, row, collisionMarksIndex, correspondingCollisionMarksIndex, row, i);
					collisionMarks[correspondingCollisionMarksIndex] = true;
					collisionsToResolve = true;
					MergeStatus mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
					switch(mergeStatus) {
						case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
					}
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__global__ 
void resolveCollidedParticles(Particle** particles, bool* collisionMarks, int n, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		//printf("resolveCollidedParticles\n");
		//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
		// CollisionResolver* collisionResolver;
		// if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
		// 	collisionResolver = new CollisionResolverCoalesce();
		// } else {
		// 	//printf("CollisionResolver could not be initialised\n");
		// 	assert(false);
		// }
		CollisionResolverCoalesce collisionResolver = CollisionResolverCoalesce();
		auto mergeStatus = mergeCollisionsRows(collisionMarks, idx, idx, n, true);
		switch(mergeStatus) {
			case LOWER_COLLISION_FOUND  ://printf("%d: LOWER_COLLISION_FOUND\n", idx);   break;
			case COLLISION_FOUND: //printf("%d: COLLISION_FOUND\n", idx); break;
			case NO_COLLISION_FOUND : //printf("%d: NO_COLLISION_FOUND\n", idx);  break;
		}
		bool collisionsToResolve = mergeStatus == COLLISION_FOUND;
		if(collisionsToResolve) {
			auto p1 = particles[idx];
			for(int i = 0; i < idx; i++) {
				int collisionMarksIndex = i + (idx-1)*idx/2;
				if (collisionMarks[collisionMarksIndex]) {
					//printf("Resolving collision: %d, %d\n", idx, i);
					auto p2 = particles[i];
					collisionResolver.resolve(p1, p2);
				}
			}
		}
		// delete collisionResolver;
	} 
}

void Collision::cpuRun(vector<Particle*>& particles)
{
	// get particles that collided
	vector<set<Particle*>*> particlesCollidedVector;
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
    	set<Particle*> particlesCollidedSet = {};
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			if (collisionDetector->isCollision(p1, p2)) {
				particlesCollidedSet.insert(p1);
				particlesCollidedSet.insert(p2);
			}
		}
		if(particlesCollidedSet.size() > 0)
			particlesCollidedVector.push_back(new set<Particle*>(particlesCollidedSet));
	}
	// merge sets of particles that collided
	for (auto it1 = particlesCollidedVector.begin(); it1 != particlesCollidedVector.end(); it1++) {
		auto particlesCollided1 = *it1;
		for (auto it2 = it1+1; it2 < particlesCollidedVector.end(); it2++) {
			auto particlesCollided2 = *it2;
			if ([&]() {
				for(auto p: *particlesCollided2) {
					if(particlesCollided1->find(p) != particlesCollided1->end()) {
						return true;
					}
				}
				return false;
			}()) {
				for(auto p: *particlesCollided2) {
					particlesCollided1->insert(p);
				}
				particlesCollided2->clear();
			}
		}
	}
	//resolve particles
	for (auto particlesCollided1: particlesCollidedVector) {
		if(particlesCollided1->size() > 0) {
			auto p1 = *(particlesCollided1->begin());
			particlesCollided1->erase(particlesCollided1->begin());
			for(auto p2: *particlesCollided1) {
				collisionResolver->resolve(p1, p2);
			}
		}
	}
	//erase particles marked for deletion safely
	for (auto it = particles.begin(); it != particles.end();) {
		if((*it)->deleted) {
			delete *it;
			it = particles.erase(it);
		}
		else
			++it;
	}
}


void Collision::gpuRun(vector<Particle*>& particles) {
	cudaWithError->setDevice(0);
	int particleCount = (int)particles.size();

	//Instantiate object on the CPU
	auto particlesArray = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i)
		particlesArray[i] = particles[i];

	//Copy dynamically allocated child objects to GPU
	Particle ** d_par;
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->malloc((void**)&d_par[i],sizeof(ParticleSimple));
		cudaWithError->memcpy(d_par[i], particlesArray[i], sizeof(ParticleSimple), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	Particle** td_par;
	cudaWithError->malloc((void**)&td_par, particleCount * sizeof(Particle *));
	cudaWithError->memcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);

	// get particles that collided
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	bool* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, betweenParticlesCount*sizeof(bool));
	getCollidedParticles <<<1 + betweenParticlesCount/256, 256>>> (td_par, collisionMarks, betweenParticlesCount, collisionDetector->getIndex());
	cudaWithError->deviceSynchronize("getCollidedParticles");

	// merge sets of particles that collided and resolve
	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (td_par, collisionMarks, particleCount, collisionResolver->getIndex());
	cudaWithError->deviceSynchronize("resolveCollidedParticles");

	//copy particles back to cpu
	for(int i = 0; i < particleCount; i++) {
		cudaWithError->memcpy(particlesArray[i],d_par[i],sizeof(ParticleSimple),hipMemcpyDeviceToHost);
		cudaWithError->free(d_par[i]);
		particles[i] = particlesArray[i];
	}

	//TODO: do this on gpu
	//erase particles marked for deletion safely
	for (auto it = particles.begin(); it != particles.end();) {
		if((*it)->deleted) {
			delete *it;
			it = particles.erase(it);
		}
		else
			++it;
	}

	cudaWithError->free(td_par);
	cudaWithError->free(collisionMarks);
	delete particlesArray;
	delete d_par;
	
	// cout << "Got to the end" << endl;
}