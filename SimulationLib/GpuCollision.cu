#include "hip/hip_runtime.h"
#include "GpuCollision.cuh"
#include "GpuCollisionHelper.cuh"
#include "ParticleSimple.h"
#include "CollisionDetectorSimple.cuh"
#include "CollisionResolverCoalesce.cuh"
#include "MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
__global__ 
void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
			collisionDetectorGpu[0] = new CollisionDetectorSimple();
		} else {
			printf("collisionDetectorGpu could not be initialised\n");
			assert(false);
		}
	} 
}
__global__ 
void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
			collisionResolverGpu[0] = new CollisionResolverCoalesce();
		} else {
			printf("collisionResolverGpu could not be initialised\n");
			assert(false);
		}
	} 
}

GpuCollision::GpuCollision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver) : GpuLaw("Collision") {
	cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
	cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
	setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
	setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
	cudaWithError->peekAtLastError("setCollisionDetector");
}

GpuCollision::~GpuCollision() {
	cudaWithError->free(collisionDetectorGpu);
	cudaWithError->free(collisionResolverGpu);
}

__global__ 
void getCollidedParticles(Particle** particles, bool* collisionMarks, CollisionDetector** collisionDetectorGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		getCollidedParticlesHelper(idx, particles, collisionMarks, collisionDetectorGpu);
	} 
}

__global__ 
void resolveCollidedParticles(Particle** particles, bool* collisionMarks, CollisionResolver** collisionResolverGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		resolveCollidedParticlesHelper(idx, particles, collisionMarks, collisionResolverGpu, n);
	} 
}

void GpuCollision::run(Particle** td_par, int particleCount) {
	// get particles that collided
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	bool* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, betweenParticlesCount*sizeof(bool));
	getCollidedParticles <<<1 + betweenParticlesCount/256, 256>>> (td_par, collisionMarks, collisionDetectorGpu, betweenParticlesCount);
	cudaWithError->peekAtLastError("getCollidedParticles");

	// merge sets of particles that collided and resolve
	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (td_par, collisionMarks, collisionResolverGpu, particleCount);
	cudaWithError->peekAtLastError("resolveCollidedParticles");

	cudaWithError->free(collisionMarks);
}