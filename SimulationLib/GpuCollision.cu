#include "hip/hip_runtime.h"
#include "GpuCollision.cuh"
#include "ParticleSimple.h"
#include "CollisionDetectorSimple.cuh"
#include "CollisionResolverCoalesce.cuh"
#include "MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
__global__ 
void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
			collisionDetectorGpu[0] = new CollisionDetectorSimple();
		} else {
			printf("collisionDetectorGpu could not be initialised\n");
			assert(false);
		}
	} 
}
__global__ 
void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
			collisionResolverGpu[0] = new CollisionResolverCoalesce();
		} else {
			printf("collisionResolverGpu could not be initialised\n");
			assert(false);
		}
	} 
}

GpuCollision::GpuCollision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver) : GpuLaw("Collision") {
	cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
	cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
	setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
	setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
	cudaWithError->peekAtLastError("setCollisionDetector");
}

GpuCollision::~GpuCollision() {
	cudaWithError->free(collisionDetectorGpu);
	cudaWithError->free(collisionResolverGpu);
}

__global__ 
void getCollidedParticles(Particle** particles, bool* collisionMarks, CollisionDetector** collisionDetectorGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		auto p1 = particles[x];
		auto p2 = particles[y];
		if (collisionDetectorGpu[0]->isCollision(p1, p2)) {
			collisionMarks[idx] = true;
		} else {
			collisionMarks[idx] = false;			
		}
	} 
}

enum MergeStatus { LOWER_COLLISION_FOUND, COLLISION_FOUND, NO_COLLISION_FOUND };

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun = false);
__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n);

__device__ MergeStatus mergeCollisionsRows(bool* collisionMarks, int idx, int row, int n, bool firstRun) {
	bool collisionsToResolve = false;
	MergeStatus mergeStatus = mergeCollisionsColumns(collisionMarks, idx, row, n);
	switch(mergeStatus) {
		case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
		case COLLISION_FOUND: collisionsToResolve = true; break;
		case NO_COLLISION_FOUND: break;
	}
	for(int i = 0; i < row; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, row);
		if(collisionMarks[collisionMarksIndex]) {
			int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
			if(firstRun || !collisionMarks[correspondingCollisionMarksIndex]) {
				collisionMarks[correspondingCollisionMarksIndex] = true;
				collisionsToResolve = true;
				mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
				switch(mergeStatus) {
					case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__device__ MergeStatus mergeCollisionsColumns(bool* collisionMarks, int idx, int row, int n) {
	bool collisionsToResolve = false;
	for(int i = row + 1; i < n; i++) {
		int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(row, i);
		if(collisionMarks[collisionMarksIndex]) {
			if(i > idx) {
				return LOWER_COLLISION_FOUND;
			} else if(i < idx) {
				int correspondingCollisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
				if(!collisionMarks[correspondingCollisionMarksIndex]) {
					collisionMarks[correspondingCollisionMarksIndex] = true;
					collisionsToResolve = true;
					MergeStatus mergeStatus = mergeCollisionsRows(collisionMarks, idx, i, n);
					switch(mergeStatus) {
						case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
					}
				}
			}
		}
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__global__ 
void resolveCollidedParticles(Particle** particles, bool* collisionMarks, CollisionResolver** collisionResolverGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		auto collisionsToResolve = mergeCollisionsRows(collisionMarks, idx, idx, n, true) == COLLISION_FOUND; 
		if(collisionsToResolve) {
			auto p1 = particles[idx];
			for(int i = 0; i < idx; i++) {
				int collisionMarksIndex = MatrixMaths::getLowerTriangularIndex(i, idx);
				if (collisionMarks[collisionMarksIndex]) {
					auto p2 = particles[i];
					collisionResolverGpu[0]->resolve(p1, p2);
				}
			}
		}
		// delete collisionResolver;
	} 
}

void GpuCollision::run(Particle** td_par, int particleCount) {
	// get particles that collided
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	bool* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, betweenParticlesCount*sizeof(bool));
	getCollidedParticles <<<1 + betweenParticlesCount/256, 256>>> (td_par, collisionMarks, collisionDetectorGpu, betweenParticlesCount);
	cudaWithError->peekAtLastError("getCollidedParticles");

	// merge sets of particles that collided and resolve
	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (td_par, collisionMarks, collisionResolverGpu, particleCount);
	cudaWithError->peekAtLastError("resolveCollidedParticles");

	cudaWithError->free(collisionMarks);
}