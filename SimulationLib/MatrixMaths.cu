#include "hip/hip_runtime.h"
#include "MatrixMaths.cuh"

#include <cmath>

__device__ __host__
void MatrixMaths::getLowerTriangularCoordinates(int i, int* x, int* y) {
    *y = ((-1+sqrt((double)8*i+1))/2) + 1;
    *x = i - (*y-1)*(*y)/2;
}

__device__ __host__
int MatrixMaths::getUpperTriangularIndex(int x, int y) {
    return y + (x-1)*x/2;
}

__device__ __host__
int MatrixMaths::getLowerTriangularIndex(int x, int y) {
    return getUpperTriangularIndex(y, x);
}