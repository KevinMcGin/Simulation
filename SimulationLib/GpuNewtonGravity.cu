#include "hip/hip_runtime.h"
#include "GpuNewtonGravity.cuh"
#include "ParticleSimple.h"
#include "Particle.cuh"
#include "MatrixMaths.cuh"
#include "NewtonGravityHeper.cuh"

#include <cmath>

GpuNewtonGravity::GpuNewtonGravity(double G) : GpuLaw("GpuNewtonGravity"), G(G) { }

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* accelerations, int n, double G) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		Vector3D devicePRadiusComponent = getRadiusComponent(particles[x], particles[y], G);
		accelerations[idx] = -getAcceleration(particles[y]->mass, devicePRadiusComponent);
		accelerations[idx + n] = getAcceleration(particles[x]->mass, devicePRadiusComponent);
	} 
}

__global__ 
void addAccelerationsKernelLower(Particle** particles, Vector3D* accelerations, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getLowerTriangularIndex(x, y);
		runOnParticle(particles[x], accelerations[radiusComponentIndex]);
	} 
}

__global__ 
void addAccelerationsKernelUpper(Particle** particles, Vector3D* accelerations, int x0, int y, int n, int betweenParticlesTriangularCount) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getUpperTriangularIndex(x, y);
		runOnParticle(particles[x], accelerations[radiusComponentIndex + betweenParticlesTriangularCount]);
	} 
}

void GpuNewtonGravity::run(Particle** td_par, int particleCount) {
	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount;
	int betweenParticlesTriangularCount = betweenParticlesCount/2;
	Vector3D* accelerations = NULL;
	cudaWithError->malloc((void**)&accelerations, betweenParticlesCount*sizeof(Vector3D));
	radiusComponentKernel <<<1 + betweenParticlesTriangularCount/256, 256>>> (td_par, accelerations, betweenParticlesTriangularCount, G);
	cudaWithError->peekAtLastError("radiusComponentKernel");

	for(int i = 0; i < particleCount; i++) {
		addAccelerationsKernelLower <<<1 + i/256, 256>>> (td_par, accelerations, 0, i, i);
		cudaWithError->peekAtLastError("addAccelerationsKernelLower");
		addAccelerationsKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, accelerations, i+1, i, particleCount, betweenParticlesTriangularCount);
		cudaWithError->peekAtLastError("addAccelerationsKernelUpper");
	}
	
	cudaWithError->free(accelerations);
}

