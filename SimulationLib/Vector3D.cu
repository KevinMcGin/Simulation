#include "hip/hip_runtime.h"
#include "Vector3D.cuh"
#include <cmath>



Vector3D::Vector3D(double x, double y, double z):
	x(x),
	y(y),
	z(z)
{

}

Vector3D::Vector3D()
{

}

Vector3D Vector3D::unit()
{
	return Vector3D(x,y,z) / this->magnitude();
}

double Vector3D::magnitudeSquared()
{
	return x * x + y * y + z * z;
}

__device__ __host__ double Vector3D::magnitude()
{
	return sqrt(this->magnitudeSquared());
}

double Vector3D::dotProduct(Vector3D vec) {
	return x * vec.x + y * vec.y + z * vec.z;
}

Vector3D Vector3D::crossProduct(Vector3D vec) {
	return {y*vec.z - vec.y*z, -(x*vec.z - vec.x*z), x*vec.y - vec.x*y};
}

__device__ __host__ Vector3D Vector3D::operator+(const Vector3D& vec) const
{
	return {
		x + vec.x,
		y + vec.y,
		z + vec.z
	};
}

Vector3D Vector3D::operator-(const Vector3D& vec) const
{
	return {
		x - vec.x,
		y - vec.y,
		z - vec.z
	};
}

Vector3D Vector3D::operator*(const double scale) const
{
	return {
		x * scale,
		y * scale,
		z * scale
	};
}



Vector3D Vector3D::operator/(const double scale) const
{
	return {
		x / scale,
		y / scale,
		z / scale
	};
}

bool Vector3D::operator==(const Vector3D& vec) const
{
	return x == vec.x && y == vec.y && z == vec.z;
}

Vector3D operator*(const double scale, const Vector3D& vec)
{
	return vec * scale;
}
