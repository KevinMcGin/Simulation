#include "hip/hip_runtime.h"
#include "Vector3D.cuh"
#include <cmath>



__device__ __host__ Vector3D::Vector3D(double x, double y, double z):
	x(x),
	y(y),
	z(z)
{

}

__device__ __host__ Vector3D::Vector3D()
{

}
__device__ __host__
 Vector3D Vector3D::unit()
{
	return Vector3D(x,y,z) / this->magnitude();
}
__device__ __host__
 double Vector3D::magnitudeSquared()
{
	return x * x + y * y + z * z;
}
__device__ __host__
 double Vector3D::magnitude()
{
	return sqrt(this->magnitudeSquared());
}
__device__ __host__
double Vector3D::dotProduct(Vector3D vec) {
	return x * vec.x + y * vec.y + z * vec.z;
}
__device__ __host__
Vector3D Vector3D::crossProduct(Vector3D vec) {
	return {y*vec.z - vec.y*z, -(x*vec.z - vec.x*z), x*vec.y - vec.x*y};
}

__device__ __host__ 
Vector3D Vector3D::operator+(const Vector3D& vec) const
{
	return {
		x + vec.x,
		y + vec.y,
		z + vec.z
	};
}
__device__ __host__
 Vector3D Vector3D::operator-(const Vector3D& vec) const
{
	return {
		x - vec.x,
		y - vec.y,
		z - vec.z
	};
}
__device__ __host__
 Vector3D Vector3D::operator*(const double scale) const
{
	return {
		x * scale,
		y * scale,
		z * scale
	};
}


__device__ __host__
 Vector3D Vector3D::operator/(const double scale) const
{
	return {
		x / scale,
		y / scale,
		z / scale
	};
}

bool Vector3D::operator==(const Vector3D& vec) const
{
	return x == vec.x && y == vec.y && z == vec.z;
}
__device__ __host__
Vector3D operator*(const double scale, const Vector3D& vec)
{
	return vec * scale;
}
