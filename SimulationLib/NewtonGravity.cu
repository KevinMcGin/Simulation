#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"

#include <cmath>



NewtonGravity::NewtonGravity() : Law(), G(PhysicalConstants::GRAVITATIONAL_CONSTANT)
{

}

NewtonGravity::NewtonGravity(double G) : Law(), G(G)
{

}


__device__ __host__ void runParticle(Particle* p1, Particle* p2, double G);	
__device__ __host__ Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G);

__global__ 
void newtonGravityKernel(CopyClass* copy, int particleMatchingsCount, double G)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleMatchingsCount) { 
		int y = (int)((-1+sqrt((float)8*idx+1))/2) + 1;
		int x = idx - (y-1)*y/2;
		printf("%d  -> (%d,%d)\n", idx, x, y);
		printf("%d -> (%lf,%lf,%lf)\n", idx, copy->par[x]->velocity.x, copy->par[x]->velocity.y, copy->par[x]->velocity.z);
		runParticle(copy->par[x],copy->par[y],G);
		printf("%d -> (%lf,%lf,%lf)\n", idx, copy->par[x]->velocity.x, copy->par[x]->velocity.y, copy->par[x]->velocity.z);
	} 
}

void NewtonGravity::run(vector<Particle*>& particles)
{
	#if USE_CUDA
		hipError_t cudaStatus;
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\ncudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}
		int particleCount = particles.size();
		int particleMatchingsCount = (particleCount-1)*particleCount/2;

		//Instantiate object on the CPU
		CopyClass cpuClass;
		cpuClass.par = new Particle*[particleCount];
		for(int i = 0; i < particleCount; ++i)
		  cpuClass.par[i] = particles[i];

		//Allocate storage for object onto GPU and copy host object to device
		CopyClass * gpuClass;
		hipMalloc(&gpuClass,sizeof(CopyClass));
		hipMemcpy(gpuClass,&cpuClass,sizeof(CopyClass),hipMemcpyHostToDevice);

		//Copy dynamically allocated child objects to GPU
		Particle ** d_par;
		d_par = new Particle*[particleCount];
		for(int i = 0; i < particleCount; ++i) {
			hipMalloc(&d_par[i],sizeof(ParticleSimple));
			printf("\tCopying data\n");
			hipMemcpy(d_par[i],cpuClass.par[i],sizeof(ParticleSimple),hipMemcpyHostToDevice);
		}

		//Copy the d_par array itself to the device

		Particle ** td_par;
		hipMalloc(&td_par, particleCount * sizeof(Particle *));
		hipMemcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);

		//copy *pointer value* of td_par to appropriate location in top level object
		hipMemcpy(&(gpuClass->par),&(td_par),sizeof(Particle **),hipMemcpyHostToDevice);

		newtonGravityKernel <<<1 + particleMatchingsCount/256, 256>>> (gpuClass, particleMatchingsCount, G);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		for(int i = 0; i < particleMatchingsCount; i++) {
			cudaStatus = hipMemcpy(cpuClass.par[i],d_par[i],sizeof(ParticleSimple),hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "\n\nNewtonGravity: cudaMemcpy4 failed: %s\n", hipGetErrorString(cudaStatus));
			}
			particles[i]->velocity = cpuClass.par[i]->velocity;
		}
	#else
		for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
			auto p1 = *it1;
			for (auto it2 = it1+1; it2 < particles.end(); it2++) {
				auto p2 = *it2;
				runParticle(p1,p2,G);
			}
		}
	#endif
}

__device__ __host__ 
void runParticle(Particle* p1, Particle* p2, double G) {	
	// printf("getting radius component");
	Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
	// printf("got radius component");
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
	Vector3D acceleration2 = p1->mass * radiusComponent * -1;
	p2->velocity = p2->velocity + acceleration2;
	// printf("updated velocities");
}

__device__ __host__ 
Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G)
{
	// printf("in getRadiusComponent");
	Vector3D displacement = position2 - position1;
	double displacementSquared = displacement.magnitudeSquared();
	// printf("if statement in getRadiusComponent");
	if (displacementSquared == 0 ) return {0,0,0};
	// printf("finished getRadiusComponent");
	return (G / displacementSquared) * displacement.unit();
}
