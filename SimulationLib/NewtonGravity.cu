#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"
#include "MatrixMaths.cuh"

#include <cmath>

NewtonGravity::NewtonGravity() : Law("NewtonGravity"), G(PhysicalConstants::GRAVITATIONAL_CONSTANT) { }

NewtonGravity::NewtonGravity(double G) : Law("NewtonGravity"), G(G) { }

void runOnParticles(Particle* p1, Particle* p2, double G);
__device__ __host__ void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent);	
__device__ __host__ Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G);

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* devicePRadiusComponent, int n, double G) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		devicePRadiusComponent[idx] = getRadiusComponent(particles[x]->position, particles[y]->position, G);
	} 
}

__global__ 
void newtonGravityKernelLower(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getLowerTriangularIndx(x, y);
		runOnParticle(particles[x], particles[y], -devicePRadiusComponent[radiusComponentIndex]);
	} 
}

__global__ 
void newtonGravityKernelUpper(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getUpperTriangularIndx(x, y);
		runOnParticle(particles[x], particles[y], devicePRadiusComponent[radiusComponentIndex]);
	} 
}

void NewtonGravity::cpuRun(vector<Particle*>& particles) {
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runOnParticles(p1,p2,G);
		}
	}
}

void NewtonGravity::gpuRun(vector<Particle*>& particles) {
	cudaWithError->setDevice(0);
	int particleCount = (int)particles.size();

	//Instantiate object on the CPU
	auto particlesArray = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i)
		particlesArray[i] = particles[i];

	//Copy dynamically allocated child objects to GPU
	Particle ** d_par;
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->malloc((void**)&d_par[i],sizeof(ParticleSimple));
		cudaWithError->memcpy(d_par[i], particlesArray[i], sizeof(ParticleSimple), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	Particle ** td_par;
	cudaWithError->malloc((void**)&td_par, particleCount * sizeof(Particle *));
	cudaWithError->memcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);

	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	Vector3D* devicePRadiusComponent = NULL;
	cudaWithError->malloc((void**)&devicePRadiusComponent, betweenParticlesCount*sizeof(Vector3D));
	radiusComponentKernel <<<1 + betweenParticlesCount/256, 256>>> (td_par, devicePRadiusComponent, betweenParticlesCount, G);
	cudaWithError->deviceSynchronize();

	for(int i = 0; i < particleCount; i++) {
		newtonGravityKernelLower <<<1 + i/256, 256>>> (td_par, devicePRadiusComponent, 0, i, i);
		newtonGravityKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, devicePRadiusComponent, i+1, i, particleCount);
		//TODO can we move this out of the loop?
		cudaWithError->deviceSynchronize();
	}
	for(int i = 0; i < particleCount; i++) {
		cudaWithError->memcpy(particlesArray[i],d_par[i],sizeof(ParticleSimple),hipMemcpyDeviceToHost);
		cudaWithError->free(d_par[i]);
		particles[i]->velocity = particlesArray[i]->velocity;
	}
	
	cudaWithError->free(devicePRadiusComponent);
	cudaWithError->free(td_par);
	delete particlesArray;
	delete d_par;
}

void runOnParticles(Particle* p1, Particle* p2, double G) {	
	Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
	runOnParticle(p1, p2, -radiusComponent);
	runOnParticle(p2, p1, radiusComponent);
}

__device__ __host__ 
void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent) {	
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
}

__device__ __host__ 
Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G)
{
	Vector3D displacement = position1 - position2;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared == 0 ) {
		return {0,0,0};
	} else {
		Vector3D unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}
