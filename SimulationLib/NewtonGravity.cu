#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"
#include "MatrixMaths.cuh"

#include <cmath>

NewtonGravity::NewtonGravity() : Law("NewtonGravity"), G(PhysicalConstants::GRAVITATIONAL_CONSTANT) { }

NewtonGravity::NewtonGravity(double G) : Law("NewtonGravity"), G(G) { }

void runOnParticles(Particle* p1, Particle* p2, double G);
__device__ __host__ void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent);	
__device__ __host__ Vector3D getRadiusComponent(Particle* p1, Particle* p2, double G);

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* devicePRadiusComponent, int n, double G) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		devicePRadiusComponent[idx] = getRadiusComponent(particles[x], particles[y], G);
	} 
}

__global__ 
void newtonGravityKernelLower(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getLowerTriangularIndx(x, y);
		runOnParticle(particles[x], particles[y], -devicePRadiusComponent[radiusComponentIndex]);
	}  
}

__global__ 
void newtonGravityKernelUpper(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getUpperTriangularIndx(x, y);
		runOnParticle(particles[x], particles[y], devicePRadiusComponent[radiusComponentIndex]);
	} 
}

void NewtonGravity::cpuRun(vector<Particle*>& particles) {
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runOnParticles(p1,p2,G);
		}
	}
} 

void NewtonGravity::gpuRun(Particle** td_par, int particleCount) {
	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	Vector3D* devicePRadiusComponent = NULL;
	cudaWithError->malloc((void**)&devicePRadiusComponent, betweenParticlesCount*sizeof(Vector3D));
	radiusComponentKernel <<<1 + betweenParticlesCount/256, 256>>> (td_par, devicePRadiusComponent, betweenParticlesCount, G);
	cudaWithError->deviceSynchronize();

	for(int i = 0; i < particleCount; i++) {
		newtonGravityKernelLower <<<1 + i/256, 256>>> (td_par, devicePRadiusComponent, 0, i, i);
		newtonGravityKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, devicePRadiusComponent, i+1, i, particleCount);
		//TODO can we move this out of the loop? 
		//Or do the calculations in parallel fully, and then apply them with this deviceSynchronize every loop
		cudaWithError->deviceSynchronize();
	}
	
	cudaWithError->free(devicePRadiusComponent);
}

void runOnParticles(Particle* p1, Particle* p2, double G) {	
	Vector3D radiusComponent = getRadiusComponent(p1, p2, G);
	runOnParticle(p1, p2, -radiusComponent);
	runOnParticle(p2, p1, radiusComponent);
}

__device__ __host__ 
void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent) {	
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
}

__device__ __host__ 
Vector3D getRadiusComponent(Particle* p1, Particle* p2, double G) {
	Vector3D displacement = p1->position - p2->position;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared <= pow(p1->radius + p2->radius, 2)) {
		return {0, 0, 0};
	} else {
		Vector3D unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}
