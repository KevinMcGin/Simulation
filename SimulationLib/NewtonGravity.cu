#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"

#include <cmath>



NewtonGravity::NewtonGravity() : Law(), G(PhysicalConstants::GRAVITATIONAL_CONSTANT)
{

}

NewtonGravity::NewtonGravity(double G) : Law(), G(G)
{

}

void runOnParticles(Particle* p1, Particle* p2, double G);
__device__ __host__ void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent);	
__device__ __host__ Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G);

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* devicePRadiusComponent, int n, double G)
{
	//printf("inside radiusComponentKernel start");
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		unsigned long y = (long long)((-1+sqrt((double)8*idx+1))/2) + 1;
		unsigned long x = idx - (y-1)*y/2;
		//printf("%lu  -> (%lu,%lu). Getting radius Component, X: (%f,%f,%f)\n", idx, x, y, particles[x]->position.x, particles[x]->position.y, particles[x]->position.z);
		// printf("%d  -> (%d,%d). Getting radius Component, Y: (%f,%f,%f)\n", idx, x, y, particles[y]->position.x, particles[y]->position.y, particles[y]->position.z);
		devicePRadiusComponent[idx] = getRadiusComponent(particles[x]->position, particles[y]->position, G);
		// printf("%d  -> (%d,%d). Got radius Component: (%f,%f,%f)\n", idx, x, y, devicePRadiusComponent[idx].x, devicePRadiusComponent[idx].y, devicePRadiusComponent[idx].z);
	} 
}

__global__ 
void newtonGravityKernelLower(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n)
{
	// printf("inside kernel lower start");
	unsigned long idx = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned long x = idx + x0;
	if(x < n) { 
		// printf("inside kernel lower");
		// printf("%f  -> (%d,%d). using radius Component, X: (%f,%f,%f)\n", x, x, y, particles[x]->position.x, particles[x]->position.y, particles[x]->position.z);
		int radiusComponentIndex = x + (y-1)*y/2;
		// printf("%d  -> (%d,%d) -> %d. using radius Component, Y: (%f,%f,%f)\n", x, x, y, radiusComponentIndex, 
		// -1*devicePRadiusComponent[radiusComponentIndex].x, -1*devicePRadiusComponent[radiusComponentIndex].y, -1*devicePRadiusComponent[radiusComponentIndex].z);
		// printf("%f  -> (%d,%d). using radius Component: (%f,%f,%f)\n", x-x0, x, y, devicePRadiusComponent[radiusComponentIndex].x, devicePRadiusComponent[radiusComponentIndex].y, devicePRadiusComponent[radiusComponentIndex].z);
		runOnParticle(particles[x], particles[y], -1*devicePRadiusComponent[radiusComponentIndex]);
	} 
}

__global__ 
void newtonGravityKernelUpper(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		// printf("%f  -> (%d,%d). using radius Component, X: (%f,%f,%f)\n", x, x, y, universe->particles[x]->position.x, universe->particles[x]->position.y, universe->particles[x]->position.z);
		int radiusComponentIndex = y + (x-1)*x/2;
		//printf("%d  -> (%d,%d) -> %d. using radius Component, Y: (%f,%f,%f)\n", x, x, y, radiusComponentIndex, 
//			devicePRadiusComponent[radiusComponentIndex].x, devicePRadiusComponent[radiusComponentIndex].y, devicePRadiusComponent[radiusComponentIndex].z);
		// printf("%f  -> (%d,%d). using radius Component: (%f,%f,%f)\n", x-x0, x, y, devicePRadiusComponent[radiusComponentIndex].x, devicePRadiusComponent[radiusComponentIndex].y, devicePRadiusComponent[radiusComponentIndex].z);
		runOnParticle(particles[x], particles[y], devicePRadiusComponent[radiusComponentIndex]);
	} 
}

void NewtonGravity::cpuRun(vector<Particle*>& particles)
{
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runOnParticles(p1,p2,G);
		}
	}
}

void NewtonGravity::gpuRun(vector<Particle*>& particles) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\ncudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	int particleCount = particles.size();
	//int particleMatchingsCount = (particleCount-1)*particleCount/2;

	//Instantiate object on the CPU
	UniverseGPU cpuClass;
	cpuClass.particles = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i)
		cpuClass.particles[i] = particles[i];

	//Copy dynamically allocated child objects to GPU
	Particle ** d_par;
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaStatus = hipMalloc(&d_par[i],sizeof(ParticleSimple));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: hipMalloc failed!\n");
		}
		cudaStatus = hipMemcpy(d_par[i],cpuClass.particles[i],sizeof(ParticleSimple),hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: hipMemcpy failed!\n");
		}
	}

	//Copy the d_par array itself to the device

	Particle ** td_par;
	cudaStatus = hipMalloc(&td_par, particleCount * sizeof(Particle *));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: hipMalloc failed!\n");
	}
	cudaStatus = hipMemcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: hipMemcpy failed!\n");
	}
	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	Vector3D* devicePRadiusComponent = NULL;
	cudaStatus = hipMalloc(&devicePRadiusComponent, betweenParticlesCount*sizeof(Vector3D));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: hipMalloc failed!\n");
	}
	radiusComponentKernel <<<1 + betweenParticlesCount/256, 256>>> (td_par, devicePRadiusComponent, betweenParticlesCount, G);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: radiusComponentKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		throw "radiusComponentKernel failed";
	}
	//std::cout << "synced device" << std::endl;

	for(int i = 0; i < particleCount; i++) {
		newtonGravityKernelLower <<<1 + i/256, 256>>> (td_par, devicePRadiusComponent, 0, i, i);
		newtonGravityKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, devicePRadiusComponent, i+1, i, particleCount);
		cudaStatus = hipGetLastError();
		hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: newtonGravityKernels launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw "newtonGravityKernels failed";
		}
		//std::cout << "synced device" << std::endl;
		cudaStatus = hipGetLastError();
	}
	for(int i = 0; i < particleCount; i++) {
		cudaStatus = hipMemcpy(cpuClass.particles[i],d_par[i],sizeof(ParticleSimple),hipMemcpyDeviceToHost);
		hipFree(d_par[i]);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\n\nNewtonGravity: hipMemcpyDeviceToHost failed: %s\n", hipGetErrorString(cudaStatus));
		}
		particles[i]->velocity = cpuClass.particles[i]->velocity;
	}
	
	hipFree(devicePRadiusComponent);
	hipFree(td_par);
	delete cpuClass.particles;
	delete d_par;
}

void runOnParticles(Particle* p1, Particle* p2, double G) {	
	Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
	runOnParticle(p1, p2, -1*radiusComponent);
	runOnParticle(p2, p1, radiusComponent);
}

//Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
__device__ __host__ 
void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent) {	
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
}

__device__ __host__ 
Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G)
{
	Vector3D displacement = position1 - position2;
	//TODO: only calculate magnitudeSquared once
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared == 0 ) return {0,0,0};
	return (G / displacementSquared) * displacement.unit();
}
