#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"
#include "MatrixMaths.cuh"

#include <cmath>

NewtonGravity::NewtonGravity() : Law("NewtonGravity"), G(PhysicalConstants::GRAVITATIONAL_CONSTANT) { }

NewtonGravity::NewtonGravity(double G) : Law("NewtonGravity"), G(G) { }

void runOnParticles(Particle* p1, Particle* p2, double G);
__device__ __host__ Vector3D getAcceleration(double mass, Vector3D radiusComponent);
__device__ __host__ void runOnParticle(Particle* p1, Vector3D acceleration);	
__device__ __host__ Vector3D getRadiusComponent(Particle* p1, Particle* p2, double G);

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* accelerations, int n, double G) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		int x, y;
		MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
		Vector3D devicePRadiusComponent = getRadiusComponent(particles[x], particles[y], G);
		accelerations[idx] = -getAcceleration(particles[y]->mass, devicePRadiusComponent);
		accelerations[idx + n] = getAcceleration(particles[x]->mass, devicePRadiusComponent);
	} 
}

__global__ 
void addAccelerationsKernelLower(Particle** particles, Vector3D* accelerations, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getLowerTriangularIndex(x, y);
		runOnParticle(particles[x], accelerations[radiusComponentIndex]);
	} 
}

__global__ 
void addAccelerationsKernelUpper(Particle** particles, Vector3D* accelerations, int x0, int y, int n, int betweenParticlesTriangularCount) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = MatrixMaths::getUpperTriangularIndex(x, y);
		runOnParticle(particles[x], accelerations[radiusComponentIndex + betweenParticlesTriangularCount]);
	} 
}

void NewtonGravity::cpuRun(vector<Particle*>& particles) {
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1 + 1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runOnParticles(p1, p2, G);
		}
	}
} 

void NewtonGravity::gpuRun(Particle** td_par, int particleCount) {
	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount;
	int betweenParticlesTriangularCount = betweenParticlesCount/2;
	Vector3D* accelerations = NULL;
	cudaWithError->malloc((void**)&accelerations, betweenParticlesCount*sizeof(Vector3D));
	radiusComponentKernel <<<1 + betweenParticlesTriangularCount/256, 256>>> (td_par, accelerations, betweenParticlesTriangularCount, G);
	cudaWithError->deviceSynchronize();

	for(int i = 0; i < particleCount; i++) {
		addAccelerationsKernelLower <<<1 + i/256, 256>>> (td_par, accelerations, 0, i, i);
		addAccelerationsKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, accelerations, i+1, i, particleCount, betweenParticlesTriangularCount);
	}
	cudaWithError->deviceSynchronize();
	
	cudaWithError->free(accelerations);
}

void runOnParticles(Particle* p1, Particle* p2, double G) {	
	Vector3D radiusComponent = getRadiusComponent(p1, p2, G);
	runOnParticle(p1, -getAcceleration(p2->mass, radiusComponent));
	runOnParticle(p2, getAcceleration(p1->mass, radiusComponent));
}

__device__ __host__ 
Vector3D getAcceleration(double mass, Vector3D radiusComponent) {	
	return mass * radiusComponent;
}

__device__ __host__ 
void runOnParticle(Particle* p1, Vector3D acceleration) {
	p1->velocity = p1->velocity + acceleration;
}

__device__ __host__ 
Vector3D getRadiusComponent(Particle* p1, Particle* p2, double G) {
	Vector3D displacement = p1->position - p2->position;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared <= pow(p1->radius + p2->radius, 2)) {
		return {0, 0, 0};
	} else {
		Vector3D unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}
