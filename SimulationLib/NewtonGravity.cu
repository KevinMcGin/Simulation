#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"

#include <cmath>



NewtonGravity::NewtonGravity() : Law(), G(PhysicalConstants::GRAVITATIONAL_CONSTANT)
{

}

NewtonGravity::NewtonGravity(double G) : Law(), G(G)
{

}

void runOnParticles(Particle* p1, Particle* p2, double G);
__device__ __host__ void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent);	
__device__ __host__ Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G);

__global__ 
void newtonGravityKernel(CopyClass* copy, int n, double G)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	if(x < n) { 
		//int y = (int)((-1+sqrt((float)8*idx+1))/2) + 1;
		 //printf("%d  -> (%d,%d)\n", x, x, n);
		// printf("%d -> (%lf,%lf,%lf)\n", idx, copy->par[x]->velocity.x, copy->par[x]->velocity.y, copy->par[x]->velocity.z);
		// runParticle(copy->par[x],copy->par[n],G);
		// printf("%d -> (%lf,%lf,%lf)\n", idx, copy->par[x]->velocity.x, copy->par[x]->velocity.y, copy->par[x]->velocity.z);
	} 
}

void NewtonGravity::run(vector<Particle*>& particles)
{
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runOnParticles(p1,p2,G);
		}
	}
}

void NewtonGravity::runParallel(vector<Particle*>& particles) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\ncudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	int particleCount = particles.size();
	//int particleMatchingsCount = (particleCount-1)*particleCount/2;

	//Instantiate object on the CPU
	CopyClass cpuClass;
	cpuClass.par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i)
		cpuClass.par[i] = particles[i];

	//Allocate storage for object onto GPU and copy host object to device
	CopyClass * gpuClass;
	hipMalloc(&gpuClass,sizeof(CopyClass));
	hipMemcpy(gpuClass,&cpuClass,sizeof(CopyClass),hipMemcpyHostToDevice);

	//Copy dynamically allocated child objects to GPU
	Particle ** d_par;
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		hipMalloc(&d_par[i],sizeof(ParticleSimple));
		hipMemcpy(d_par[i],cpuClass.par[i],sizeof(ParticleSimple),hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device

	Particle ** td_par;
	hipMalloc(&td_par, particleCount * sizeof(Particle *));
	hipMemcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);

	//copy *pointer value* of td_par to appropriate location in top level object
	hipMemcpy(&(gpuClass->par),&(td_par),sizeof(Particle **),hipMemcpyHostToDevice);

	for(int i = 0; i < particleCount; i++) {
		newtonGravityKernel <<<1 + i/256, 256>>> (gpuClass, i, G);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
	}
	for(int i = 0; i < particleCount; i++) {
		cudaStatus = hipMemcpy(cpuClass.par[i],d_par[i],sizeof(ParticleSimple),hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\n\nNewtonGravity: hipMemcpyDeviceToHost failed: %s\n", hipGetErrorString(cudaStatus));
		}
		particles[i]->velocity = cpuClass.par[i]->velocity;
	}
}

void runOnParticles(Particle* p1, Particle* p2, double G) {	
	Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
	runOnParticle(p1, p2, -1*radiusComponent);
	runOnParticle(p2, p1, radiusComponent);
}

//Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
__device__ __host__ 
void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent) {	
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
}

__device__ __host__ 
Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G)
{
	Vector3D displacement = position1 - position2;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared == 0 ) return {0,0,0};
	return (G / displacementSquared) * displacement.unit();
}
