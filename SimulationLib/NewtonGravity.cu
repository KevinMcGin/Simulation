#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include <cmath>



NewtonGravity::NewtonGravity() : Law(), G(PhysicalConstants::GRAVITATIONAL_CONSTANT)
{

}

NewtonGravity::NewtonGravity(double G) : Law(), G(G)
{

}

__global__ 
static void newtonGravityKernel(Particle** particles, int particleMatchingsCount)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleMatchingsCount) { 
		//runParticle(p1,p2);
	} 
}

void NewtonGravity::run(vector<Particle*>& particles)
{
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runParticle(p1,p2);
		}
	}

}

__device__ __host__ 
void NewtonGravity::runParticle(Particle* p1, Particle* p2) {	
	Vector3D displacement = (p2->position - p1->position);
	double displacementSquared = displacement.magnitudeSquared();
	Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position);
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
	Vector3D acceleration2 = p1->mass * radiusComponent * -1;
	p2->velocity = p2->velocity + acceleration2;
}

__device__ __host__ 
Vector3D NewtonGravity::getRadiusComponent(Vector3D position1, Vector3D position2)
{
	Vector3D displacement = position2 - position1;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared == 0 ) return {0,0,0};
	return (G / displacementSquared) * displacement.unit();
}
