#include "hip/hip_runtime.h"
#include "NewtonGravity.cuh"
#include "ParticleSimple.h"

#include <cmath>



NewtonGravity::NewtonGravity() : Law("NewtonGravity"), G(PhysicalConstants::GRAVITATIONAL_CONSTANT) { }

NewtonGravity::NewtonGravity(double G) : Law("NewtonGravity"), G(G) { }

void runOnParticles(Particle* p1, Particle* p2, double G);
__device__ __host__ void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent);	
__device__ __host__ Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G);

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* devicePRadiusComponent, int n, double G) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		unsigned long y = (long long)((-1+sqrt((double)8*idx+1))/2) + 1;
		unsigned long x = idx - (y-1)*y/2;
		devicePRadiusComponent[idx] = getRadiusComponent(particles[x]->position, particles[y]->position, G);
	} 
}

__global__ 
void newtonGravityKernelLower(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n) {
	unsigned long idx = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned long x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = x + (y-1)*y/2;
		runOnParticle(particles[x], particles[y], -devicePRadiusComponent[radiusComponentIndex]);
	} 
}

__global__ 
void newtonGravityKernelUpper(Particle** particles, Vector3D* devicePRadiusComponent, int x0, int y, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int x = idx + x0;
	if(x < n) { 
		int radiusComponentIndex = y + (x-1)*x/2;
		runOnParticle(particles[x], particles[y], devicePRadiusComponent[radiusComponentIndex]);
	} 
}

void NewtonGravity::cpuRun(vector<Particle*>& particles) {
	for (auto it1 = particles.begin(); it1 != particles.end(); it1++) {
		auto p1 = *it1;
		for (auto it2 = it1+1; it2 < particles.end(); it2++) {
			auto p2 = *it2;
			runOnParticles(p1,p2,G);
		}
	}
}

void NewtonGravity::gpuRun(vector<Particle*>& particles) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\ncudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	int particleCount = particles.size();

	//Instantiate object on the CPU
	UniverseGPU cpuClass;
	cpuClass.particles = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i)
		cpuClass.particles[i] = particles[i];

	//Copy dynamically allocated child objects to GPU
	Particle ** d_par;
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaStatus = hipMalloc(&d_par[i],sizeof(ParticleSimple));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: hipMalloc failed!\n");
		}
		cudaStatus = hipMemcpy(d_par[i], cpuClass.particles[i], sizeof(ParticleSimple), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: hipMemcpy failed!\n");
		}
	}

	//Copy the d_par array itself to the device
	Particle ** td_par;
	cudaStatus = hipMalloc(&td_par, particleCount * sizeof(Particle *));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: hipMalloc failed!\n");
	}
	cudaStatus = hipMemcpy(td_par, d_par, particleCount * sizeof(Particle *), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: hipMemcpy failed!\n");
	}
	//Radius component
	int betweenParticlesCount = (particleCount-1)*particleCount/2;
	Vector3D* devicePRadiusComponent = NULL;
	cudaStatus = hipMalloc(&devicePRadiusComponent, betweenParticlesCount*sizeof(Vector3D));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: hipMalloc failed!\n");
	}
	radiusComponentKernel <<<1 + betweenParticlesCount/256, 256>>> (td_par, devicePRadiusComponent, betweenParticlesCount, G);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonGravity: radiusComponentKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		throw "radiusComponentKernel failed";
	}

	for(int i = 0; i < particleCount; i++) {
		newtonGravityKernelLower <<<1 + i/256, 256>>> (td_par, devicePRadiusComponent, 0, i, i);
		newtonGravityKernelUpper <<<1 + (particleCount-1-i)/256, 256>>> (td_par, devicePRadiusComponent, i+1, i, particleCount);
		cudaStatus = hipGetLastError();
		hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\nNewtonGravity: newtonGravityKernels launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw "newtonGravityKernels failed";
		}
		cudaStatus = hipGetLastError();
	}
	for(int i = 0; i < particleCount; i++) {
		cudaStatus = hipMemcpy(cpuClass.particles[i],d_par[i],sizeof(ParticleSimple),hipMemcpyDeviceToHost);
		hipFree(d_par[i]);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\n\nNewtonGravity: hipMemcpyDeviceToHost failed: %s\n", hipGetErrorString(cudaStatus));
		}
		particles[i]->velocity = cpuClass.particles[i]->velocity;
	}
	
	hipFree(devicePRadiusComponent);
	hipFree(td_par);
	delete cpuClass.particles;
	delete d_par;
}

void runOnParticles(Particle* p1, Particle* p2, double G) {	
	Vector3D radiusComponent = getRadiusComponent(p1->position, p2->position, G);
	runOnParticle(p1, p2, -1*radiusComponent);
	runOnParticle(p2, p1, radiusComponent);
}

__device__ __host__ 
void runOnParticle(Particle* p1, Particle* p2, Vector3D radiusComponent) {	
	Vector3D acceleration1 = p2->mass * radiusComponent;
	p1->velocity = p1->velocity + acceleration1;
}

__device__ __host__ 
Vector3D getRadiusComponent(Vector3D position1, Vector3D position2, double G)
{
	Vector3D displacement = position1 - position2;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared == 0 ) {
		return {0,0,0};
	} else {
		Vector3D unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}
