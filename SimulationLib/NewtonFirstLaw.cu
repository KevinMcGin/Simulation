#include "hip/hip_runtime.h"
#include "NewtonFirstLaw.cuh"
#include "Particle.cuh"

__global__ 
static void advanceParticles(Vector3D* position, const Vector3D* velocity, int particleCount)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleCount) { 
		position[idx] = position[idx] + velocity[idx];
	} 
}

void NewtonFirstLaw::run(vector<Particle*>& particles) {
	for (const auto& p : particles)
		p->advance();
}

void NewtonFirstLaw::runParallel(vector<Particle*>& particles) {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	int n = particles.size();
	Vector3D* pPosition = new Vector3D[n];
	Vector3D* pVelocity = new Vector3D[n];
	for(int i = 0; i < n; i++) {
		pPosition[i] = particles[i]->position;
		pVelocity[i] = particles[i]->velocity;
	}
	Vector3D* devicePPosition = NULL;
	Vector3D* devicePVelocity = NULL;
	hipMalloc(&devicePPosition, n*sizeof(Vector3D));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: hipMalloc failed!\n");
	}
	hipMalloc(&devicePVelocity, n*sizeof(Vector3D));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: hipMalloc failed!\n");
	}
	cudaStatus = hipMemcpy(devicePPosition, pPosition, n*sizeof(Vector3D), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: hipMemcpy failed!\n");
	}
	cudaStatus = hipMemcpy(devicePVelocity, pVelocity, n*sizeof(Vector3D), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: hipMemcpy failed!\n");
	}
	advanceParticles <<<1 + n/256, 256>>> (devicePPosition, devicePVelocity, n);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipMemcpy(pPosition, devicePPosition, n*sizeof(Vector3D), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\nNewtonFirstLaw: hipMemcpy failed!\n");
	}

	for(int i = 0; i < n; i++)
		particles[i]->position = pPosition[i];
	
	hipFree(devicePPosition);
	hipFree(devicePVelocity);
}
