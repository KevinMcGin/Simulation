#include "hip/hip_runtime.h"
#include "NewtonFirstLaw.cuh"
#include "Particle.cuh"

__global__ 
static void advanceParticles(Vector3D* position, const Vector3D* velocity, int particleCount)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleCount) { 
		position[idx] = position[idx] + velocity[idx];
	} 
}

void NewtonFirstLaw::run(vector<Particle*>& particles)
{
	#ifdef USE_CUDA
		hipError_t cudaStatus;
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}
		int n = particles.size();
		Vector3D* pPosition = new Vector3D[n];
		Vector3D* pVelocity = new Vector3D[n];
		for(int i = 0; i < n; i++) {
			pPosition[i] = particles[i]->position;
			pVelocity[i] = particles[i]->velocity;
		}
		Vector3D* devicePPosition = NULL;
		Vector3D* devicePVelocity = NULL;
		hipMalloc(&devicePPosition, n*sizeof(Vector3D));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: hipMalloc failed!");
		}
		hipMalloc(&devicePVelocity, n*sizeof(Vector3D));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(devicePPosition, pPosition, n*sizeof(Vector3D), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: hipMemcpy failed!");
		}
		cudaStatus = hipMemcpy(devicePVelocity, pVelocity, n*sizeof(Vector3D), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: hipMemcpy failed!");
		}
		advanceParticles <<<1 + n/256, 256>>> (devicePPosition, devicePVelocity, n);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		cudaStatus = hipMemcpy(pPosition, devicePPosition, n*sizeof(Vector3D), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "NewtonFirstLaw: hipMemcpy failed!");
		}
		for(int i = 0; i < n; i++)
			particles[i]->position = pPosition[i];
	#else
		for (const auto& p : particles)
			p->advance();
	#endif
}
