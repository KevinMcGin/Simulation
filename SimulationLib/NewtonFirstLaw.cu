#include "hip/hip_runtime.h"
#include "NewtonFirstLaw.h"
#include "Particle.cuh"

__global__ 
void advanceParticles(Vector3D* position, const Vector3D* velocity, int particleCount)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleCount) { 
		position[idx].x = position[idx].x + velocity[idx].x;
		position[idx].y = position[idx].y + velocity[idx].y;
		position[idx].z = position[idx].z + velocity[idx].z;
		//particles[idx]->advance();
	} 
}

void NewtonFirstLaw::run(vector<Particle*>& particles)
{
	#ifdef USE_CUDA
		hipError_t cudaStatus;
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		}
		int n = particles.size();
		Vector3D* pPosition = new Vector3D[n];
		Vector3D* pVelocity = new Vector3D[n];
		for(int i = 0; i < n; i++) {
			pPosition[i] = particles[i]->position;
			pVelocity[i] = particles[i]->velocity;
		}
		Vector3D* devicePPosition = NULL;
		Vector3D* devicePVelocity = NULL;
		hipMalloc(&devicePPosition, n*sizeof(Vector3D));
		hipMalloc(&devicePVelocity, n*sizeof(Vector3D));
		cudaStatus = hipMemcpy(devicePPosition, pPosition, n*sizeof(Vector3D), hipMemcpyHostToDevice);
		cudaStatus = hipMemcpy(devicePVelocity, pVelocity, n*sizeof(Vector3D), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		advanceParticles <<<1 + n/256, 256>>> (devicePPosition, devicePVelocity, n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		cudaStatus = hipMemcpy(pPosition, devicePPosition, n*sizeof(Vector3D), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		for(int i = 0; i < n; i++)
			particles[i]->position = pPosition[i];
	#else
		for (const auto& p : particles)
			p->advance();
	#endif
}
