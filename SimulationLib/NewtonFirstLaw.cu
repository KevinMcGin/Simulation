#include "hip/hip_runtime.h"
#include "NewtonFirstLaw.cuh"
#include "Particle.cuh"

NewtonFirstLaw::NewtonFirstLaw() : Law("NewtonFirstLaw") { }


__global__ 
static void advanceParticles(Particle** particles, int particleCount) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleCount) { 
		particles[idx]->position = particles[idx]->position + particles[idx]->velocity;
	} 
}

void NewtonFirstLaw::cpuRun(vector<Particle*>& particles) {
	for (const auto& p : particles)
		p->advance();
}

void NewtonFirstLaw::gpuRun(Particle** td_par, int particleCount) {
	advanceParticles <<<1 + particleCount/256, 256>>> (td_par, particleCount);
	cudaWithError->deviceSynchronize();
}
