#include "hip/hip_runtime.h"
#include "NewtonFirstLaw.cuh"
#include "Particle.cuh"

NewtonFirstLaw::NewtonFirstLaw() : Law("NewtonFirstLaw") { }


__global__ 
static void advanceParticles(Vector3D* position, const Vector3D* velocity, int particleCount)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleCount) { 
		position[idx] = position[idx] + velocity[idx];
	} 
}

void NewtonFirstLaw::cpuRun(vector<Particle*>& particles) {
	for (const auto& p : particles)
		p->advance();
}

void NewtonFirstLaw::gpuRun(vector<Particle*>& particles) {
	cudaWithError->setDevice(0);
	int n = particles.size();
	Vector3D* pPosition = new Vector3D[n];
	Vector3D* pVelocity = new Vector3D[n];
	for(int i = 0; i < n; i++) {
		pPosition[i] = particles[i]->position;
		pVelocity[i] = particles[i]->velocity;
	}
	Vector3D* devicePPosition = NULL;
	Vector3D* devicePVelocity = NULL;
	cudaWithError->malloc((void**)&devicePPosition, n*sizeof(Vector3D));
	cudaWithError->malloc((void**)&devicePVelocity, n*sizeof(Vector3D));
	cudaWithError->memcpy(devicePPosition, pPosition, n*sizeof(Vector3D), hipMemcpyHostToDevice);
	cudaWithError->memcpy(devicePVelocity, pVelocity, n*sizeof(Vector3D), hipMemcpyHostToDevice);
	advanceParticles <<<1 + n/256, 256>>> (devicePPosition, devicePVelocity, n);
	cudaWithError->deviceSynchronize();
	cudaWithError->memcpy(pPosition, devicePPosition, n*sizeof(Vector3D), hipMemcpyDeviceToHost);

	for(int i = 0; i < n; i++)
		particles[i]->position = pPosition[i];
	
	cudaWithError->free(devicePPosition);
	cudaWithError->free(devicePVelocity);
}
