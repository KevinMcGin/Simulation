#include "hip/hip_runtime.h"
#include "CollisionDetectorSimple.cuh"

__device__ __host__
bool CollisionDetectorSimple::isCollision(Particle* p1, Particle* p2)
{
	Vector3D difference = p1->position - p2->position;
	double absoluteDifference = sqrt(pow(difference.x, 2) + pow(difference.y, 2) + pow(difference.z, 2));
	return absoluteDifference < p1->radius + p2->radius;
}
