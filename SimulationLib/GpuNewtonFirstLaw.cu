#include "hip/hip_runtime.h"
#include "GpuNewtonFirstLaw.cuh"
#include "Particle.cuh"

GpuNewtonFirstLaw::GpuNewtonFirstLaw() : GpuLaw("NewtonFirstLaw") { }

__global__ 
static void advanceParticles(Particle** particles, int particleCount) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < particleCount) { 
		particles[idx]->advance();
	} 
}

void GpuNewtonFirstLaw::run(Particle** td_par, int particleCount) {
	advanceParticles <<<1 + particleCount/256, 256>>> (td_par, particleCount);
	cudaWithError->peekAtLastError("advanceParticles");
}
