#include "hip/hip_runtime.h"
#include "util/MatrixMaths.cuh"

#include <cmath>
#include< cstdio>
using namespace std;

__device__ __host__
void MatrixMaths::getLowerTriangularCoordinates(unsigned long long i, unsigned long long* x, unsigned long long* y) {
    *y = ((-1+sqrt((double)8*i+1))/2) + 1;
    *x = i - (*y-1)*(*y)/2;
}

__device__ __host__
unsigned long long MatrixMaths::getUpperTriangularIndex(unsigned long long x, unsigned long long y) {
    return y + (x-1)*x/2;
}

__device__ __host__
unsigned long long MatrixMaths::getLowerTriangularIndex(unsigned long long x, unsigned long long y) {
    return getUpperTriangularIndex(y, x);
}