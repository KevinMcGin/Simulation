#include "hip/hip_runtime.h"
#include "util/MatrixMaths.cuh"

#include <cmath>

#if defined(USE_GPU)
   __device__ __host__
#endif
void MatrixMaths::getLowerTriangularCoordinates(unsigned long long i, unsigned long long* x, unsigned long long* y) {
    *y = (unsigned long long)(((-1+sqrt((double)8*i+1))/2) + 1);
    *x = i - (*y-1)*(*y)/2;
}

#if defined(USE_GPU)
   __device__ __host__
#endif
unsigned long long MatrixMaths::getUpperTriangularIndex(unsigned long long x, unsigned long long y) {
    return y + (x-1)*x/2;
}

#if defined(USE_GPU)
   __device__ __host__
#endif
unsigned long long MatrixMaths::getLowerTriangularIndex(unsigned long long x, unsigned long long y) {
    return getUpperTriangularIndex(y, x);
}