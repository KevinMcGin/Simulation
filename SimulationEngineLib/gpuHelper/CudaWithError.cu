#include "hip/hip_runtime.h"
#include "gpuHelper/CudaWithError.cuh"
#include <iostream>
#include <algorithm>

unsigned long long CudaWithError::minMemoryRemaining = 0;
unsigned long long CudaWithError::maxMemoryPerEvent = ULLONG_MAX;

CudaWithError::CudaWithError(std::string className): className(className) { }

void CudaWithError::throwErrorMaybe(hipError_t cudaStatus, std::string error) {
    if (cudaStatus != hipSuccess) {
        std::string totalError = className + ": " + error + "\n" + hipGetErrorString(cudaStatus);
        std::cerr << std::endl << totalError << std::endl;
        throw std::runtime_error(totalError);
    }
}

void CudaWithError::setDevice(int device) {
	hipError_t cudaStatus = hipSetDevice(device);
    throwErrorMaybe(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
}

void CudaWithError::malloc(void** devPtr, size_t size) {
    hipError_t cudaStatus = hipMalloc(devPtr, size);
    throwErrorMaybe(cudaStatus, "hipMalloc failed!");
}

void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {
    hipError_t cudaStatus = hipMemcpy(dst, src, count, kind);
    throwErrorMaybe(cudaStatus, "hipMemcpy failed!");
}

void CudaWithError::deviceSynchronize(std::string message) {
    hipError_t cudaStatus = hipDeviceSynchronize();
    throwErrorMaybe(cudaStatus, message + ": hipDeviceSynchronize failed!");
}

void CudaWithError::free(void* devPtr) {
    hipError_t cudaStatus = hipFree(devPtr);
    throwErrorMaybe(cudaStatus, "hipFree failed!");
}

void CudaWithError::peekAtLastError(std::string message) {
    hipError_t cudaStatus = hipPeekAtLastError();
    throwErrorMaybe(cudaStatus, message + ": hipPeekAtLastError failed!");
}

unsigned long long CudaWithError::getFreeGpuMemory() {
    size_t free_t, total_t;
    hipMemGetInfo(&free_t, &total_t);
    float factor = 1.0485760;
    auto freeMemory = (unsigned long long)(free_t / factor) - CudaWithError::minMemoryRemaining;
    return std::min(freeMemory, maxMemoryPerEvent); 
}

unsigned long long CudaWithError::getMaxThreads() {
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    return ((unsigned long long)properties.multiProcessorCount) * ((unsigned long long)properties.maxThreadsPerMultiProcessor);
}

void CudaWithError::setMinMemoryRemaining(unsigned long long minMemoryRemaining) {
    CudaWithError::minMemoryRemaining = minMemoryRemaining;
}

void CudaWithError::setMaxMemoryPerEvent(unsigned long long maxMemoryPerEvent) {
    CudaWithError::maxMemoryPerEvent = maxMemoryPerEvent;
}