#include "gpuHelper/CudaWithError.cuh"
#include<iostream>


CudaWithError::CudaWithError(std::string className): className(className) { }

void CudaWithError::throwErrorMaybe(hipError_t cudaStatus, std::string error) {
    if (cudaStatus != hipSuccess) {
        std::string totalError = className + ": " + error + "\n" + hipGetErrorString(cudaStatus);
        std::cerr << std::endl << totalError << std::endl;
        throw new std::runtime_error(totalError);
    }
}

void CudaWithError::setDevice(int device) {
	hipError_t cudaStatus = hipSetDevice(device);
    throwErrorMaybe(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
}

void CudaWithError::malloc(void** devPtr, size_t size) {
    hipError_t cudaStatus = hipMalloc(devPtr, size);
    throwErrorMaybe(cudaStatus, "hipMalloc failed!");
}

void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {
    hipError_t cudaStatus = hipMemcpy(dst, src, count, kind);
    throwErrorMaybe(cudaStatus, "hipMemcpy failed!");
}

void CudaWithError::deviceSynchronize(std::string message) {
    hipError_t cudaStatus = hipDeviceSynchronize();
    throwErrorMaybe(cudaStatus, message + ": hipDeviceSynchronize failed!");
}

void CudaWithError::free(void* devPtr) {
    hipError_t cudaStatus = hipFree(devPtr);
    throwErrorMaybe(cudaStatus, "hipFree failed!");
}

void CudaWithError::peekAtLastError(std::string message) {
    hipError_t cudaStatus = hipPeekAtLastError();
    throwErrorMaybe(cudaStatus, message + ": hipPeekAtLastError failed!");
}


unsigned long CudaWithError::getFreeGpuMemory() 
{
    size_t free_t, total_t;

    hipMemGetInfo(&free_t, &total_t);

    free_t = free_t / 1.0485760;

    unsigned long free_m = (unsigned int)free_t;

    return free_m;
}
