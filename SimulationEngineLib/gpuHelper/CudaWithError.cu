#include "hip/hip_runtime.h"
#include "gpuHelper/CudaWithError.cuh"
#include <iostream>
#include <algorithm>

#if defined(USE_GPU)
    unsigned long long CudaWithError::minMemoryRemaining = 0;
    unsigned long long CudaWithError::maxMemoryPerEvent = ULLONG_MAX;
    unsigned long CudaWithError::kernelSize = 256;
#endif

CudaWithError::CudaWithError(std::string className): className(className) { }

#if defined(USE_GPU)
    void CudaWithError::throwErrorMaybe(hipError_t cudaStatus, std::string error) {
        if (cudaStatus != hipSuccess) {
            std::string totalError = className + ": " + error + "\n" + hipGetErrorString(cudaStatus);
            std::cerr << std::endl << totalError << std::endl;
            throw std::runtime_error(totalError);
        }
    }

    void CudaWithError::setDevice(int device) {
        hipError_t cudaStatus = hipSetDevice(device);
        throwErrorMaybe(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    void CudaWithError::malloc(void** devPtr, size_t size) {
        hipError_t cudaStatus = hipMalloc(devPtr, size);
        throwErrorMaybe(cudaStatus, "hipMalloc failed!");
    }

    void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {
        hipError_t cudaStatus = hipMemcpy(dst, src, count, kind);
        throwErrorMaybe(cudaStatus, "hipMemcpy failed!");
    }

    void CudaWithError::deviceSynchronize(std::string message) {
        hipError_t cudaStatus = hipDeviceSynchronize();
        throwErrorMaybe(cudaStatus, message + ": hipDeviceSynchronize failed!");
    }

    void CudaWithError::free(void* devPtr) {
        hipError_t cudaStatus = hipFree(devPtr);
        throwErrorMaybe(cudaStatus, "hipFree failed!");
    }

    void CudaWithError::peekAtLastError(std::string message) {
        hipError_t cudaStatus = hipPeekAtLastError();
        throwErrorMaybe(cudaStatus, message + ": hipPeekAtLastError failed!");
    }

    unsigned long long CudaWithError::getFreeGpuMemory() {
        size_t free_t, total_t;
        hipMemGetInfo(&free_t, &total_t);
        float factor = 1.0485760;
        auto freeMemory = (unsigned long long)(free_t / factor) - CudaWithError::minMemoryRemaining;
        return std::min(freeMemory, maxMemoryPerEvent); 
    }

    unsigned long long CudaWithError::getMaxThreads() {
        struct hipDeviceProp_t properties;
        hipGetDeviceProperties(&properties, 0);
        return ((unsigned long long)properties.multiProcessorCount) * ((unsigned long long)properties.maxThreadsPerMultiProcessor);
    }

    void CudaWithError::runKernel(std::string message, std::function<void (unsigned int kernelSize)> kernelMethod) {
        kernelMethod(CudaWithError::kernelSize);
		peekAtLastError(message);
    }

    void CudaWithError::setMinMemoryRemaining(unsigned long long minMemoryRemaining) {
        CudaWithError::minMemoryRemaining = minMemoryRemaining;
    }

    void CudaWithError::setMaxMemoryPerEvent(unsigned long long maxMemoryPerEvent) {
        CudaWithError::maxMemoryPerEvent = maxMemoryPerEvent;
    }

    void CudaWithError::setKernelSize(unsigned long kernelSize) {
        CudaWithError::kernelSize = kernelSize;
    }
#else

    void CudaWithError::throwErrorMaybe(hipError_t cudaStatus, std::string error) {}

    void CudaWithError::setDevice(int device) {}

    void CudaWithError::malloc(void** devPtr, size_t size) {}

    void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {}

    void CudaWithError::deviceSynchronize(std::string message) {}

    void CudaWithError::free(void* devPtr) {}

    void CudaWithError::peekAtLastError(std::string message) {}

    unsigned long long CudaWithError::getFreeGpuMemory() { return 0; }

    unsigned long long CudaWithError::getMaxThreads() { return 0; }

    void CudaWithError::runKernel(std::string message, std::function<void (unsigned int kernelSize)> kernelMethod) {}

    void CudaWithError::setMinMemoryRemaining(unsigned long long minMemoryRemaining) {}

    void CudaWithError::setMaxMemoryPerEvent(unsigned long long maxMemoryPerEvent) {}

    void CudaWithError::setKernalSize(unsigned long long kernalSize) {}
#endif
