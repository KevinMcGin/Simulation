#include "gpuHelper/GpuDataController.cuh"
#include "particle/ParticleSimple.h"

GpuDataController::GpuDataController() {
    cudaWithError = new CudaWithError("GpuDataController");
    cudaWithError->setDevice(0);
}

GpuDataController::~GpuDataController() {
    delete cudaWithError;
	for(int i = 0; i < particleCount; i++) {
		cudaWithError->free(d_par[i]);
	}
	cudaWithError->free(td_par);
	delete d_par;
}

void GpuDataController::putParticlesOnDevice(vector<Particle*> particles, bool firstRun) {
	if(!firstRun) {
		for(int i = 0; i < particleCount; i++) {
			cudaWithError->free(d_par[i]);
		}
		cudaWithError->free(td_par);
		delete d_par;
	}
	
	particleCount = (int)particles.size();

	//Copy dynamically allocated child objects to GPU
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->malloc((void**)&d_par[i], sizeof(*particles[i]));
		cudaWithError->memcpy(d_par[i], particles[i], sizeof(*particles[i]), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	cudaWithError->malloc((void**)&td_par, particleCount * sizeof(Particle*));
	cudaWithError->memcpy(td_par, d_par, particleCount * sizeof(Particle*), hipMemcpyHostToDevice);
}

void GpuDataController::getParticlesFromDevice(vector<Particle*>& particles) {
	cudaWithError->deviceSynchronize();
	for(int i = 0; i < particleCount; i++) {
		cudaWithError->memcpy(particles[i], d_par[i], sizeof(*particles[i]), hipMemcpyDeviceToHost);
	}
}

