#include "gpuMock/gpuHelper/CudaWithError.cuh"
#include <iostream>
#include <algorithm>


CudaWithError::CudaWithError(std::string className) { }

void CudaWithError::setDevice(int device) {}

void CudaWithError::resetDevice() {}

void CudaWithError::malloc(void** devPtr, size_t size) {}

void CudaWithError::memcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {}

void CudaWithError::deviceSynchronize(std::string message) {}

void CudaWithError::free(void* devPtr) {}

void CudaWithError::peekAtLastError(std::string message) {}

unsigned long long CudaWithError::getFreeGpuMemory() { return 0; }

unsigned long long CudaWithError::getMaxThreads() { return 0; }

void CudaWithError::runKernel(std::string message, std::function<void (unsigned int kernelSize)> kernelMethod) {}

void CudaWithError::setMinMemoryRemaining(unsigned long long minMemoryRemaining) {}

void CudaWithError::setMaxMemoryPerEvent(unsigned long long maxMemoryPerEvent) {}

void CudaWithError::setKernelSize(unsigned long kernelSize) {}