#include "hip/hip_runtime.h"
#include "law/collision/gpuKernel/GpuCollision.cuh"
#include "law/collision/helper/GpuCollisionHelper.cuh"
#include "particle/ParticleSimple.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"
#include "law/collision/resolver/CollisionResolverCoalesce.cuh"
#include "util/MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
__global__ 
void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
			collisionDetectorGpu[0] = new CollisionDetectorSimple();
		} else {
			printf("collisionDetectorGpu could not be initialised\n");
			assert(false);
		}
	} 
}
__global__ 
void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
			collisionResolverGpu[0] = new CollisionResolverCoalesce();
		} else {
			printf("collisionResolverGpu could not be initialised\n");
			assert(false);
		}
	} 
} 

GpuCollision::GpuCollision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver) : GpuLaw("Collision") {
	cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
	cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
	setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
	setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
	cudaWithError->peekAtLastError("setCollisionDetector");
}

GpuCollision::~GpuCollision() {
	cudaWithError->free(collisionDetectorGpu);
	cudaWithError->free(collisionResolverGpu);
}

__global__ 
void getCollidedParticles(Particle** particles, unsigned long long betweenParticlesOffset, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionDetector** collisionDetectorGpu, unsigned long long n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		getCollidedParticlesHelper(idx + betweenParticlesOffset, particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionDetectorGpu);
	} 
}

__global__ 
void resolveCollidedParticles(Particle** particles, int* collisionMarks,  unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionResolver** collisionResolverGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		resolveCollidedParticlesHelper(idx, particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionResolverGpu, n);
	} 
}

void GpuCollision::run(Particle** particles, int particleCount) {
	unsigned long long betweenParticlesPairsCount = ((unsigned long long)particleCount-1)*particleCount;
	unsigned long long betweenParticlesCount = betweenParticlesPairsCount/2;

	unsigned long long freeGpuMemory = cudaWithError->getFreeGpuMemory();
	unsigned long long particlesCollidedSize = particleCount * sizeof(bool);
	unsigned long long maxIntsAllocatableStage1 = (freeGpuMemory - particlesCollidedSize - sizeof(unsigned long long)) / sizeof(int);
	unsigned long long maxIntsAllocatableFactor = 200;
	unsigned long long maxIntsAllocatable = std::min(maxIntsAllocatableStage1, betweenParticlesPairsCount * maxIntsAllocatableFactor);
	// cout << maxIntsAllocatable << endl;
	int* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, maxIntsAllocatable * sizeof(int));

	unsigned long long* collisionMarksIndex = NULL;
	cudaWithError->malloc((void**)&collisionMarksIndex, sizeof(unsigned long long));
	unsigned long long collisionMarksIndexDefault = 0;
	cudaWithError->memcpy(collisionMarksIndex, &collisionMarksIndexDefault, sizeof(unsigned long long), hipMemcpyHostToDevice);
	
	bool* particlesCollided = NULL;
	bool particlesCollidedDefault = false;
	cudaWithError->malloc((void**)&particlesCollided, particleCount * sizeof(bool));
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->memcpy(&(particlesCollided[i]), &particlesCollidedDefault, sizeof(bool), hipMemcpyHostToDevice);
	}

	//TODO this may need to be configured based on the number of threads and or timeout time
	const unsigned long long maxBetweenParticlesPerGet = 1 * 1000 * 1000;
	for(unsigned long long betweenParticlesOffset = 0; betweenParticlesOffset < betweenParticlesCount; betweenParticlesOffset += maxBetweenParticlesPerGet) {
		const unsigned long long thisBetweenParticlesCount = std::min(maxBetweenParticlesPerGet, betweenParticlesCount - betweenParticlesOffset);
		getCollidedParticles <<<1 + thisBetweenParticlesCount/256, 256>>> (particles, betweenParticlesOffset, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionDetectorGpu, thisBetweenParticlesCount);
		cudaWithError->peekAtLastError("getCollidedParticles");
	}

	//TODO remove deviceSynchronize
	cudaWithError->deviceSynchronize("get");

	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionResolverGpu, particleCount);
	cudaWithError->peekAtLastError("resolveCollidedParticles");

	//TODO remove deviceSynchronize
	cudaWithError->deviceSynchronize("resolved");
	cudaWithError->free(collisionMarks);
	cudaWithError->free(collisionMarksIndex);
	cudaWithError->free(particlesCollided);
}