#include "hip/hip_runtime.h"
#include "law/collision/gpuKernel/GpuCollision.cuh"
#include "law/collision/helper/GpuCollisionHelper.cuh"
#include "particle/ParticleSimple.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"
#include "law/collision/resolver/CollisionResolverCoalesce.cuh"
#include "util/MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
__global__ 
void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
			collisionDetectorGpu[0] = new CollisionDetectorSimple();
		} else {
			printf("collisionDetectorGpu could not be initialised\n");
			assert(false);
		}
	} 
}
__global__ 
void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
			collisionResolverGpu[0] = new CollisionResolverCoalesce();
		} else {
			printf("collisionResolverGpu could not be initialised\n");
			assert(false);
		}
	} 
} 

GpuCollision::GpuCollision(std::shared_ptr<CollisionDetector> collisionDetector, std::shared_ptr<CollisionResolver> collisionResolver) : 
	GpuLaw("Collision") {
	cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
	cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
	setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
	setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
	cudaWithError->peekAtLastError("setCollisionDetector");
}

GpuCollision::~GpuCollision() {
	cudaWithError->free(collisionDetectorGpu);
	cudaWithError->free(collisionResolverGpu);
}

__global__ 
void getCollidedParticles(Particle** particles, unsigned long long betweenParticlesOffset, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionDetector** collisionDetectorGpu, unsigned long long n) {
	unsigned long long idx = threadIdx.x + (unsigned long long)blockIdx.x*blockDim.x;
	if(idx < n) { 
		getCollidedParticlesHelper(idx + betweenParticlesOffset, particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionDetectorGpu);
	} 
}

__global__ 
void resolveCollidedParticles(Particle** particles, int particlesOffset, int* collisionMarks,  unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionResolver** collisionResolverGpu, int thisParticleCount, int particleCount) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < thisParticleCount) { 
		resolveCollidedParticlesHelper(idx + particlesOffset, particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionResolverGpu, particleCount);
	} 
}

void GpuCollision::run(Particle** particles, int particleCount) {
	unsigned long long betweenParticlesPairsCount = ((unsigned long long)particleCount-1)*particleCount;
	unsigned long long betweenParticlesCount = betweenParticlesPairsCount/2;

	long long freeGpuMemory = cudaWithError->getFreeGpuMemory();
	long long particlesCollidedSize = particleCount * sizeof(bool);
	long long maxIntsAllocatableStage1 = (freeGpuMemory - particlesCollidedSize - (long long)sizeof(unsigned long long)) / (long long)sizeof(int);
	unsigned long long maxIntsAllocatableFactor = 200;
	long long maxIntsAllocatable = std::min(maxIntsAllocatableStage1, (long long)(betweenParticlesPairsCount * maxIntsAllocatableFactor));
	if(maxIntsAllocatable <= 0) {
		throw std::runtime_error("Ran out of GPU memory");
	}
	int* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, maxIntsAllocatable * sizeof(int));

	unsigned long long* collisionMarksIndex = NULL;
	cudaWithError->malloc((void**)&collisionMarksIndex, sizeof(unsigned long long));
	
	bool* particlesCollided = NULL;
	cudaWithError->malloc((void**)&particlesCollided, particleCount * sizeof(bool));

	const unsigned long long maxThreads = cudaWithError->getMaxThreads();
	const unsigned long long maxBetweenParticlesPerGetOverMaxThreads = 37;
	const unsigned long long maxBetweenParticlesPerGet = maxThreads * maxBetweenParticlesPerGetOverMaxThreads;
	const unsigned long long maxParticlesPerResolveOverMaxThreads = 37;
	const unsigned int maxParticlesPerResolve = (unsigned int)(maxThreads * maxBetweenParticlesPerGetOverMaxThreads);
	unsigned long long collisionMarksIndexCpu = 0;
	const unsigned int maxLoops = 20;
	unsigned int indexLoops = 0;
	do {
		if(++indexLoops > maxLoops) {
			std::cout << "Max Loops in GpuCollision reached" << std::endl;
			throw std::runtime_error("Max Loops in GpuCollision reached");
		}
		collisionMarksIndexCpu = 0;
		cudaWithError->memcpy(collisionMarksIndex, &collisionMarksIndexCpu, sizeof(collisionMarksIndexCpu), hipMemcpyHostToDevice);
		bool particlesCollidedDefault = false;
		for(int particleIndex = 0; particleIndex < particleCount; ++particleIndex) {
			cudaWithError->memcpy(&(particlesCollided[particleIndex]), &particlesCollidedDefault, sizeof(bool), hipMemcpyHostToDevice);
		}

		for(unsigned long long betweenParticlesOffset = 0; betweenParticlesOffset < betweenParticlesCount; betweenParticlesOffset += maxBetweenParticlesPerGet) {
			const unsigned long long thisBetweenParticlesCount = std::min(maxBetweenParticlesPerGet, betweenParticlesCount - betweenParticlesOffset);
			getCollidedParticles <<<1 + thisBetweenParticlesCount/256, 256>>> (particles, betweenParticlesOffset, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionDetectorGpu, thisBetweenParticlesCount);
			cudaWithError->peekAtLastError("getCollidedParticles");
		}
		cudaWithError->deviceSynchronize("getCollidedParticles");

		for(int particlesOffset = 0; particlesOffset < particleCount; particlesOffset += maxParticlesPerResolve) {
			const int thisParticleCount = std::min(maxParticlesPerResolve, (unsigned int)(particleCount - particlesOffset));
			resolveCollidedParticles <<<1 + thisParticleCount/256, 256>>> (particles, particlesOffset, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionResolverGpu, thisParticleCount, particleCount);
			cudaWithError->peekAtLastError("resolveCollidedParticles");
		}
		cudaWithError->deviceSynchronize("resolveCollidedParticles");
		cudaWithError->memcpy(&collisionMarksIndexCpu, collisionMarksIndex, sizeof(collisionMarksIndexCpu), hipMemcpyDeviceToHost);
	} while(collisionMarksIndexCpu >= maxIntsAllocatable - 1);

	cudaWithError->free(collisionMarks);
	cudaWithError->free(collisionMarksIndex);
	cudaWithError->free(particlesCollided);
}