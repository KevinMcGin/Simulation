#include "hip/hip_runtime.h"
#include "law/collision/gpuKernel/GpuCollision.cuh"
#include "law/collision/helper/GpuCollisionHelper.cuh"
#include "particle/ParticleSimple.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"
#include "law/collision/resolver/CollisionResolverCoalesce.cuh"
#include "util/MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
__global__ 
void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
			collisionDetectorGpu[0] = new CollisionDetectorSimple();
		} else {
			printf("collisionDetectorGpu could not be initialised\n");
			assert(false);
		}
	} 
}
__global__ 
void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < 1) {
		if(collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
			collisionResolverGpu[0] = new CollisionResolverCoalesce();
		} else {
			printf("collisionResolverGpu could not be initialised\n");
			assert(false);
		}
	} 
} 

GpuCollision::GpuCollision(CollisionDetector* collisionDetector, CollisionResolver* collisionResolver) : GpuLaw("Collision") {
	cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
	cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
	setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
	setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
	cudaWithError->peekAtLastError("setCollisionDetector");
}

GpuCollision::~GpuCollision() {
	cudaWithError->free(collisionDetectorGpu);
	cudaWithError->free(collisionResolverGpu);
}

__global__ 
void getCollidedParticles(Particle** particles, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionDetector** collisionDetectorGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		getCollidedParticlesHelper(idx, particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionDetectorGpu);
	} 
}

__global__ 
void resolveCollidedParticles(Particle** particles, int* collisionMarks,  unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionResolver** collisionResolverGpu, int n) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		resolveCollidedParticlesHelper(idx, particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionResolverGpu, n);
	} 
}

void GpuCollision::run(Particle** particles, int particleCount) {
	// get particles that collided
	unsigned long long betweenParticlesCount = ((unsigned long long)particleCount-1)*particleCount/2;

	unsigned long long freeGpuMemory = cudaWithError->getFreeGpuMemory();
	unsigned long long particlesCollidedSize = particleCount * sizeof(bool);
	unsigned long long maxIntsAllocatableStage1 = (freeGpuMemory - particlesCollidedSize - sizeof(unsigned long long)) / sizeof(int);
	unsigned long long maxIntsAllocatable = std::min(maxIntsAllocatableStage1, betweenParticlesCount * 2 * 2000);
	// cout << maxIntsAllocatable << endl;
	int* collisionMarks = NULL;
	cudaWithError->malloc((void**)&collisionMarks, maxIntsAllocatable * sizeof(int));

	unsigned long long* collisionMarksIndex = NULL;
	cudaWithError->malloc((void**)&collisionMarksIndex, sizeof(unsigned long long));
	unsigned long long collisionMarksIndexDefault = 0;
	cudaWithError->memcpy(collisionMarksIndex, &collisionMarksIndexDefault, sizeof(unsigned long long), hipMemcpyHostToDevice);
	
	bool* particlesCollided = NULL;
	bool particlesCollidedDefault = false;
	cudaWithError->malloc((void**)&particlesCollided, particleCount * sizeof(bool));
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError->memcpy(&(particlesCollided[i]), &particlesCollidedDefault, sizeof(bool), hipMemcpyHostToDevice);
	}

	getCollidedParticles <<<1 + betweenParticlesCount/256, 256>>> (particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionDetectorGpu, betweenParticlesCount);
	cudaWithError->peekAtLastError("getCollidedParticles");

	//TODO remove deviceSynchronize
	cudaWithError->deviceSynchronize("get");

	// merge sets of particles that collided and resolve
	resolveCollidedParticles <<<1 + particleCount/256, 256>>> (particles, collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, collisionResolverGpu, particleCount);
	cudaWithError->peekAtLastError("resolveCollidedParticles");

	//TODO remove deviceSynchronize
	cudaWithError->deviceSynchronize("resolved");
	cudaWithError->free(collisionMarks);
	cudaWithError->free(collisionMarksIndex);
	cudaWithError->free(particlesCollided);
}