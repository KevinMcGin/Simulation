#include "hip/hip_runtime.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"

__device__ __host__
bool CollisionDetectorSimple::isCollision(Particle* p1, Particle* p2)
{
	Vector3D<float> difference = p1->position - p2->position;
	float magnitudeSquared = difference.magnitudeSquared();
	return magnitudeSquared < pow(p1->radius + p2->radius, 2);
}
