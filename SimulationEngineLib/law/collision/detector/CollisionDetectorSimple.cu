#include "hip/hip_runtime.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"

#if defined(USE_GPU)
   __device__ __host__
#endif
bool CollisionDetectorSimple::isCollision(Particle* p1, Particle* p2)
{
	Vector3D<float> difference = p1->position - p2->position;
	float magnitudeSquared = difference.magnitudeSquared();
	return magnitudeSquared < pow(p1->radius + p2->radius, 2);
}
