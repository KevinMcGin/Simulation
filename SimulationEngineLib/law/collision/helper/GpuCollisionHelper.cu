#include "hip/hip_runtime.h"
#include "law/collision/helper/GpuCollisionHelper.cuh"
#include "particle/ParticleSimple.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"
#include "law/collision/resolver/CollisionResolverCoalesce.cuh"
#include "util/MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

const int MAX_MERGE_COLLISION_RUNS = 5;
const int MAX_COLLISIONS_PER_PARTICLE = 100;

#if defined(USE_GPU) 
	__device__
#endif
MergeStatus mergeCollisionsRows(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int particleIndex1, int particleIndex2, int particleCount, int runCount = 0);
#if defined(USE_GPU) 
	__device__
#endif
MergeStatus mergeCollisionsColumns(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable,  bool* particlesCollided, int particleIndex1, int particleIndex2, int particleCount, int runCount = 0);

#if defined(USE_GPU) 
	__device__
#endif
int getNextParticleCollidedByIndex(int* collisionMarks, unsigned long long collisionMarksIndex, unsigned long long* thisCollisionMarksIndex, int particleIndex, bool index1 = true) {
	unsigned long long indexOffset1 = index1 ? 0 : 1;
	unsigned long long indexOffset2 = index1 ? 1 : 0;
	unsigned long long thisCollisionMarksIndexActual = *thisCollisionMarksIndex;
	while(thisCollisionMarksIndexActual < collisionMarksIndex) {
		unsigned long long index1Long = thisCollisionMarksIndexActual + indexOffset1;
		unsigned long long index2Long = thisCollisionMarksIndexActual + indexOffset2;
		int thisParticleIndex1 = collisionMarks[index1Long];
		int thisParticleIndex2 = collisionMarks[index2Long];
		thisCollisionMarksIndexActual += 2;
		if (thisParticleIndex1 == particleIndex) {
			*thisCollisionMarksIndex = thisCollisionMarksIndexActual;
			return thisParticleIndex2;
		}
		
	}
	*thisCollisionMarksIndex = thisCollisionMarksIndexActual;
	return -1;
}

#if defined(USE_GPU) 
	__device__
#endif
bool getParticlesCollidedByIndex(int* collisionMarks, unsigned long long collisionMarksIndex, int particleIndex1, int particleIndex2) {
	unsigned long long thisCollisionMarksIndexActual = 0;
	int thisParticleIndex2 = getNextParticleCollidedByIndex(collisionMarks, collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex1);
	while(thisParticleIndex2 >= 0) {
		if (thisParticleIndex2 == particleIndex2) {
			return true;
		}
		thisParticleIndex2 = getNextParticleCollidedByIndex(collisionMarks, collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex1);
	}
	return false;
}

#if defined(USE_GPU) 
	__device__
	bool markCollision(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int particleIndex1, int particleIndex2) {
		unsigned long long i = atomicAdd(collisionMarksIndex, 2);
		if (i + 1 >= maxIntsAllocatable - 1) {
			return false;
		}
		particlesCollided[particleIndex1] = true;
		collisionMarks[i] = particleIndex1;
		collisionMarks[i + 1] = particleIndex2;
		return true;
	}
#endif

#if defined(USE_GPU) 
	__device__
	MergeStatus mergeCollisionsRows(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int particleIndex1, int particleIndex2, int particleCount, int runCount) {  
		int collisionsFound = 0;
		if (!particlesCollided[particleIndex2] || (runCount > 0 && particleIndex1 == particleIndex2) || runCount >= MAX_MERGE_COLLISION_RUNS) {
			return NO_COLLISION_FOUND;
		}
		bool collisionsToResolve = false;
		MergeStatus mergeStatus = mergeCollisionsColumns(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, particleIndex1, particleIndex2, particleCount, runCount + 1);
		switch(mergeStatus) {
			case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
			case COLLISION_FOUND: collisionsToResolve = true; break;
			case NO_COLLISION_FOUND: break;
		}	
		unsigned long long thisCollisionMarksIndexActual = 0;
		int collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex2);
		while(collidedParticleIndex >= 0) {
			if (runCount == 0 || !getParticlesCollidedByIndex(collisionMarks, *collisionMarksIndex, particleIndex1, collidedParticleIndex)) {
				if (++collisionsFound >= MAX_COLLISIONS_PER_PARTICLE) {
					return COLLISION_FOUND;
				}
				if (runCount > 0 && particleIndex1 != collidedParticleIndex) {
					bool collisionMarked = markCollision(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, particleIndex1, collidedParticleIndex);
					if (!collisionMarked) {
						return NO_COLLISION_FOUND;
					}
				}
				collisionsToResolve = true;
				mergeStatus = mergeCollisionsRows(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, particleIndex1, collidedParticleIndex, particleCount, runCount + 1);
				switch(mergeStatus) {
					case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
				}
			}
			collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex2);
		}
		return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
	}
#endif

#if defined(USE_GPU) 
	__device__
	MergeStatus mergeCollisionsColumns(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int particleIndex1, int particleIndex2, int particleCount, int runCount) {
		int collisionsFound = 0;
		if (runCount >= MAX_MERGE_COLLISION_RUNS) {
			return NO_COLLISION_FOUND;
		}
		bool collisionsToResolve = false;
		unsigned long long thisCollisionMarksIndexActual = 0;
		int collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex2, false);
		while(collidedParticleIndex >= 0) {
			if (collidedParticleIndex > particleIndex1) {
				return LOWER_COLLISION_FOUND;
			} else if (collidedParticleIndex < particleIndex1) {
				if (!getParticlesCollidedByIndex(collisionMarks, *collisionMarksIndex, particleIndex1, collidedParticleIndex)) {
					if (++collisionsFound >= MAX_COLLISIONS_PER_PARTICLE) {
						return COLLISION_FOUND;
					}
					bool collisionMarked = markCollision(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, particleIndex1, collidedParticleIndex);
					if (!collisionMarked) {
						return NO_COLLISION_FOUND;
					}
					collisionsToResolve = true;
					MergeStatus mergeStatus = mergeCollisionsRows(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, particleIndex1, collidedParticleIndex, particleCount, runCount + 1);
					switch(mergeStatus) {
						case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
					}
				}
			}
			collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex2, false);
		}
		return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
	}
#endif

#if defined(USE_GPU) 
	__device__
#endif

void resolveCollidedParticlesHelper(int particleIndex, Particle** particles, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionResolver** collisionResolverGpu, int particleCount) {
	if (particlesCollided[particleIndex]) {
		auto collisionsToResolve = mergeCollisionsRows(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, particleIndex, particleIndex, particleCount) == COLLISION_FOUND; 
		if (collisionsToResolve) {
			auto p1 = particles[particleIndex];
			for(unsigned long long i = 0; i < *collisionMarksIndex; i += 2) {
				if (collisionMarks[i] == particleIndex) {
					int particleCollidedIndex = collisionMarks[i + 1];
					auto p2 = particles[particleCollidedIndex];
					if (p1->particlesExist(p2)) {
						(*collisionResolverGpu)->resolve(p1, p2);
					}
				}
			}
		}
	}
}

#if defined(USE_GPU) 
	__device__
	void getCollidedParticlesHelper(unsigned long long lowerTriangularIndex, Particle** particles, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionDetector** collisionDetectorGpu) {
		unsigned long long x, y;
		MatrixMaths::getLowerTriangularCoordinates(lowerTriangularIndex, &x, &y);
		auto p1 = particles[x];
		auto p2 = particles[y];	
		if (p1->particlesExist(p2) && (*collisionDetectorGpu)->isCollision(p1, p2)) {		
			bool collisionMarked = markCollision(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, (int)y, (int)x);
			if (!collisionMarked) {
				printf("getCollidedParticlesHelper: collisionMarks overflow\n");
			}
		}
	}
#endif