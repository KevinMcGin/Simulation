#include "hip/hip_runtime.h"
#include "law/collision/helper/GpuCollisionHelper.cuh"
#include "particle/ParticleSimple.h"
#include "law/collision/detector/CollisionDetectorSimple.cuh"
#include "law/collision/resolver/CollisionResolverCoalesce.cuh"
#include "util/MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

const int MAX_MERGE_COLLISION_RUNS = 5;

__device__ MergeStatus mergeCollisionsRows(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int idx, int row, int n, int runCount = 0);
__device__ MergeStatus mergeCollisionsColumns(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable,  bool* particlesCollided, int idx, int row, int n, int runCount = 0);

__device__ int getNextParticleCollidedByIndex(int* collisionMarks, unsigned long long collisionMarksIndex, unsigned long long* thisCollisionMarksIndex, int particleIndex, bool index1 = true) {
	unsigned long long indexOffset1 = index1 ? 0 : 1;
	unsigned long long indexOffset2 = index1 ? 1 : 0;
	unsigned long long thisCollisionMarksIndexActual = *thisCollisionMarksIndex;
	while(thisCollisionMarksIndexActual < collisionMarksIndex) {
		unsigned long long index1Long = thisCollisionMarksIndexActual + indexOffset1;
		unsigned long long index2Long = thisCollisionMarksIndexActual + indexOffset2;
		int thisParticleIndex1 = collisionMarks[index1Long];
		int thisParticleIndex2 = collisionMarks[index2Long];
		// printf("particle indices: %llu: %d, %d\n", thisCollisionMarksIndexActual, thisParticleIndex1, thisParticleIndex2);
		thisCollisionMarksIndexActual += 2;
		// printf("getNextParticleCollidedByIndex: %llu, %llu\n", index1Long, index2Long);
		if(thisParticleIndex1 == particleIndex) {
			// printf("getNextParticleCollidedByIndex: %llu, %llu\n", index1Long, index2Long);
			*thisCollisionMarksIndex = thisCollisionMarksIndexActual;
			return thisParticleIndex2;
		}
		
	}
	*thisCollisionMarksIndex = thisCollisionMarksIndexActual;
	return -1;
}

__device__ bool getParticlesCollidedByIndex(int* collisionMarks, unsigned long long collisionMarksIndex, int particleIndex1, int particleIndex2) {
	unsigned long long thisCollisionMarksIndexActual = 0;
	int thisParticleIndex2 = getNextParticleCollidedByIndex(collisionMarks, collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex1);
	while(thisParticleIndex2 >= 0) {
		//printf("thisParticleIndex2: %d, %d\n", particleIndex1, thisParticleIndex2);
		if(thisParticleIndex2 == particleIndex2) {
			return true;
		}
		thisParticleIndex2 = getNextParticleCollidedByIndex(collisionMarks, collisionMarksIndex, &thisCollisionMarksIndexActual, particleIndex1);
	}
	return false;
}

__device__ void markCollision(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int particleIndex1, int particleIndex2) {
	unsigned long long i = atomicAdd(collisionMarksIndex, 2);
	// printf("maxIntsAllocatable: %llu\n", maxIntsAllocatable);
	// printf("Marking collision at: %llu: %d, %d\n", i, particleIndex1, particleIndex2);
	if(i + 1 >= maxIntsAllocatable - 1) {
		printf("GpuCollisionHelper: collisionMarks overflow\n");
		return;
	}
	particlesCollided[particleIndex1] = true;
	collisionMarks[i] = particleIndex1;
	collisionMarks[i + 1] = particleIndex2;
}

__device__
MergeStatus mergeCollisionsRows(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int idx, int row, int n, int runCount) {  
	// printf("rows: idx: %d row: %d\n", idx, row);
	if(!particlesCollided[row] || (runCount > 0 && idx == row) || runCount >= MAX_MERGE_COLLISION_RUNS) {
		return NO_COLLISION_FOUND;
	}
	bool collisionsToResolve = false;
	MergeStatus mergeStatus = mergeCollisionsColumns(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, idx, row, n, runCount + 1);
	//printf("row: %d, mergeStatus: %d\n", row, mergeStatus);
	switch(mergeStatus) {
		case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
		case COLLISION_FOUND: collisionsToResolve = true; break;
		case NO_COLLISION_FOUND: break;
	}	
	unsigned long long thisCollisionMarksIndexActual = 0;
	int collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, row);
	//printf("collidedParticleIndex: %d\n", collidedParticleIndex);
	while(collidedParticleIndex >= 0) {
		if(runCount == 0 || !getParticlesCollidedByIndex(collisionMarks, *collisionMarksIndex, idx, collidedParticleIndex)) {
			//printf("rows: collidedParticleIndex: %d\n", collidedParticleIndex);
			if(runCount > 0 && idx != collidedParticleIndex) {
				// printf("rows: collisionMarksIndex: %llu\n", *collisionMarksIndex);
				markCollision(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, idx, collidedParticleIndex);
			}
			collisionsToResolve = true;
			mergeStatus = mergeCollisionsRows(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, idx, collidedParticleIndex, n, runCount + 1);
			switch(mergeStatus) {
				case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
			}
		}
		collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, row);
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__device__ 
MergeStatus mergeCollisionsColumns(int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, int idx, int row, int n, int runCount) {
	if(runCount >= MAX_MERGE_COLLISION_RUNS) {
		return NO_COLLISION_FOUND;
	}
	//printf("cols: idx: %d, row: %d\n", idx, row);
	bool collisionsToResolve = false;
	unsigned long long thisCollisionMarksIndexActual = 0;
	int collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, row, false);
	//printf("cols: idx: %d, row: %d, collidedParticleIndex: %d\n", idx, row, collidedParticleIndex);
	while(collidedParticleIndex >= 0) {
		if(collidedParticleIndex > idx) {
			return LOWER_COLLISION_FOUND;
		} else if(collidedParticleIndex < idx) {
			if(!getParticlesCollidedByIndex(collisionMarks, *collisionMarksIndex, idx, collidedParticleIndex)) {
				//printf("cols: %d, %d\n", idx, collidedParticleIndex);
				//printf("cols: collisionMarksIndex: %llu\n", *collisionMarksIndex);
				// printf("col: Marking collision at: %llu: %d, %d\n", *collisionMarksIndex, idx, collidedParticleIndex);
				markCollision(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, idx, collidedParticleIndex);
				collisionsToResolve = true;
				MergeStatus mergeStatus = mergeCollisionsRows(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, idx, collidedParticleIndex, n, runCount + 1);
				switch(mergeStatus) {
					case LOWER_COLLISION_FOUND: return LOWER_COLLISION_FOUND;
				}
			}
		}
		collidedParticleIndex = getNextParticleCollidedByIndex(collisionMarks, *collisionMarksIndex, &thisCollisionMarksIndexActual, row, false);
	}
	return collisionsToResolve ? COLLISION_FOUND : NO_COLLISION_FOUND;
}

__device__ 
void resolveCollidedParticlesHelper(int idx, Particle** particles, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionResolver** collisionResolverGpu, int n) {
	if(particlesCollided[idx]) {
		auto collisionsToResolve = mergeCollisionsRows(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, idx, idx, n) == COLLISION_FOUND; 
		if(collisionsToResolve) {
			// printf("resolving particle: %d\n", idx);
			auto p1 = particles[idx];
			for(unsigned long long i = 0; i < *collisionMarksIndex; i += 2) {
				//printf("resolveCollidedParticlesHelper: collisionMarksIndex: %llu\n", i);
				if(collisionMarks[i] == idx) {
					int particleCollidedIndex = collisionMarks[i + 1];
					//printf("Resolving: %d - %d\n", idx, particleCollidedIndex);
					auto p2 = particles[particleCollidedIndex];
					(*collisionResolverGpu)->resolve(p1, p2);
				}
			}
		}
	}
}

__device__
void getCollidedParticlesHelper(unsigned long long idx, Particle** particles, int* collisionMarks, unsigned long long* collisionMarksIndex, unsigned long long maxIntsAllocatable, bool* particlesCollided, CollisionDetector** collisionDetectorGpu) {
	unsigned long long x, y;
	MatrixMaths::getLowerTriangularCoordinates(idx, &x, &y);
	//TODO find out why out of bounds error is occuring here
	auto p1 = particles[x];
	auto p2 = particles[y];	
	if((*collisionDetectorGpu)->isCollision(p1, p2)) {		
		// //printf("Collision: %llu - %llu\n", x, y);	
		// //printf("Getting collision: %llu \n", *collisionMarksIndex);
		// printf("get: Marking collision at: %llu: %d, %d\n", *collisionMarksIndex, (int)y, (int)x);
		markCollision(collisionMarks, collisionMarksIndex, maxIntsAllocatable, particlesCollided, (int)y, (int)x);
	}
}