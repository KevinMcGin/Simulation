#include "hip/hip_runtime.h"
#include "law/gravity/gpuKernel/GpuNewtonGravity.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "law/gravity/helper/GpuNewtonGravityHelper.cuh"

#include <cmath>
#include <algorithm>

GpuNewtonGravity::GpuNewtonGravity(float G) : GpuLaw("GpuNewtonGravity"), G(G) { }

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D<float>* accelerations, unsigned long long n, float G, unsigned long long vectorsProcessedTriangular) {
	unsigned long long idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		radiusComponentKernelHelper(idx, particles, accelerations, n, G, vectorsProcessedTriangular);
	} 
}

__global__ 
void addAccelerationsKernelLower(Particle** particles, Vector3D<float>* accelerations, unsigned long long y, unsigned long long vectorsProcessedTriangular) {
	unsigned long long idx = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelLowerHelper(idx, particles, accelerations, y, vectorsProcessedTriangular);
}

__global__ 
void addAccelerationsKernelUpper(Particle** particles, Vector3D<float>* accelerations, unsigned long long xOffset, unsigned long long y, unsigned long long n, unsigned long long vectorsProcessedTriangular, unsigned long long betweenParticlesTriangularCount) {
	unsigned long long idx = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelUpperHelper(idx, particles, accelerations, xOffset, y, n, vectorsProcessedTriangular, betweenParticlesTriangularCount);
}

unsigned long long getRowsFromRowsAndColsCountMinusIdentity(unsigned long long rowsAndColsCount) {
	return 0.5 * (sqrt(4 * rowsAndColsCount + 1) + 1);
}

unsigned long long getRowsAndColsCountMinusIdentityFromRows(unsigned long long rowsAndColsCountMinusIdentityowsAndColsCount) {
	return (rowsAndColsCountMinusIdentityowsAndColsCount - 1) * rowsAndColsCountMinusIdentityowsAndColsCount;
}

void GpuNewtonGravity::run(Particle** td_par, int particleCount) {
	unsigned long long betweenParticlesCount = ((unsigned long long)particleCount-1)*particleCount;
	Vector3D<float>* accelerations = NULL;
	
	long long freeGpuMemory = cudaWithError->getFreeGpuMemory();
	unsigned long long vector3DSize = sizeof(Vector3D<float>);
	long long maxVectorsAllocatableStage1 = freeGpuMemory / vector3DSize;
	long long maxVectorsAllocatable = std::min(maxVectorsAllocatableStage1, (long long)betweenParticlesCount);
	if(maxVectorsAllocatable <= 0) {
		throw std::runtime_error("Ran out of GPU memory");
	}

	cudaWithError->malloc((void**)&accelerations, maxVectorsAllocatable * sizeof(Vector3D<float>));

	unsigned long long particlesProcessed = 0;
	unsigned long long vectorsProcessed = 0;
	while(particlesProcessed < particleCount) {
		unsigned long long particlesProcessable = std::min(
			getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed + maxVectorsAllocatable),
			(unsigned long long)particleCount
		) - (vectorsProcessed > 0 ? getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed) : 0);
		if(particlesProcessable == 0) { 
			std::cout << "GPU can not run these many particles in Gravity\n";
			throw std::runtime_error("GPU can not run these many particles in Gravity");
		}
		unsigned long long vectorsProcessable = getRowsAndColsCountMinusIdentityFromRows(particlesProcessed + particlesProcessable) - getRowsAndColsCountMinusIdentityFromRows(particlesProcessed);
		unsigned long long vectorsProcessableTriangular = vectorsProcessable / 2;
		const unsigned int threadCount = 256;

		radiusComponentKernel <<<1 + vectorsProcessableTriangular/threadCount, threadCount>>> (td_par, accelerations, vectorsProcessableTriangular, G, vectorsProcessed / 2);
		cudaWithError->peekAtLastError("radiusComponentKernel");

		for(int i = 0; i < particlesProcessed; i++) {
			addAccelerationsKernelUpper <<<1 + particlesProcessable / threadCount, threadCount>>> (
				td_par, accelerations, std::max((int)particlesProcessed - i - 1, 0), i, particlesProcessed + particlesProcessable, vectorsProcessed / 2, vectorsProcessableTriangular
			);
			cudaWithError->peekAtLastError("addAccelerationsKernelUpper");
		}
		for(int i = particlesProcessed; i < particlesProcessable + particlesProcessed; i++) {
			addAccelerationsKernelLower <<<1 + i/threadCount, threadCount>>> (td_par, accelerations, i, vectorsProcessed / 2);
			cudaWithError->peekAtLastError("addAccelerationsKernelLower");
			addAccelerationsKernelUpper <<<1 + particlesProcessable / threadCount, threadCount>>> (
				td_par, accelerations, 0, i, particlesProcessed + particlesProcessable, vectorsProcessed / 2, vectorsProcessableTriangular
			);
			cudaWithError->peekAtLastError("addAccelerationsKernelUpper");
		}

		particlesProcessed += particlesProcessable;
		vectorsProcessed += vectorsProcessable;
	}
	
	cudaWithError->free(accelerations);
}

