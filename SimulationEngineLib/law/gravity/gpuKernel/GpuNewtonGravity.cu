#include "hip/hip_runtime.h"
#include "law/gravity/gpuKernel/GpuNewtonGravity.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "law/gravity/helper/GpuNewtonGravityHelper.cuh"

#include <cmath>
#include <algorithm>

GpuNewtonGravity::GpuNewtonGravity(double G) : GpuLaw("GpuNewtonGravity"), G(G) { }

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D* accelerations, unsigned long long n, double G, unsigned long long vectorsProcessedTriangular) {
	unsigned long long idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < n) { 
		radiusComponentKernelHelper(idx, particles, accelerations, n, G, vectorsProcessedTriangular);
	} 
}

__global__ 
void addAccelerationsKernelLower(Particle** particles, Vector3D* accelerations, unsigned long long x0, unsigned long long y, unsigned long long n, unsigned long long vectorsProcessedTriangular) {
	unsigned long long idx = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelLowerHelper(idx, particles, accelerations, x0, y, n, vectorsProcessedTriangular);
}

__global__ 
void addAccelerationsKernelUpper(Particle** particles, Vector3D* accelerations, unsigned long long x0, unsigned long long y, unsigned long long n, unsigned long long vectorsProcessedTriangular, unsigned long long particlesProcessed, unsigned long long betweenParticlesTriangularCount) {
	unsigned long long idx = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelUpperHelper(idx, particles, accelerations, x0, y, n, vectorsProcessedTriangular, particlesProcessed, betweenParticlesTriangularCount);
}

unsigned long long getRowsFromRowsAndColsCountMinusIdentity(unsigned long long rowsAndColsCount) {
	return 0.5 * (sqrt(4 * rowsAndColsCount + 1) + 1);
}

unsigned long long getRowsAndColsCountMinusIdentityFromRows(unsigned long long rowsAndColsCountMinusIdentityowsAndColsCount) {
	return (rowsAndColsCountMinusIdentityowsAndColsCount - 1) * rowsAndColsCountMinusIdentityowsAndColsCount;
}

void GpuNewtonGravity::run(Particle** td_par, int particleCount) {
	//Radius component
	unsigned long long betweenParticlesCount = ((unsigned long long)particleCount-1)*particleCount;
	Vector3D* accelerations = NULL;
	
	unsigned long long freeGpuMemory = cudaWithError->getFreeGpuMemory();
	unsigned long long vector3DSize = sizeof(Vector3D);
	unsigned long long maxVectorsAllocatableStage1 = freeGpuMemory / vector3DSize;
	unsigned long long maxVectorsAllocatable = std::min(maxVectorsAllocatableStage1, betweenParticlesCount);

	cudaWithError->malloc((void**)&accelerations, maxVectorsAllocatable * sizeof(Vector3D));

	unsigned long long particlesProcessed = 0;
	unsigned long long vectorsProcessed = 0;
	while(particlesProcessed < particleCount) {
		unsigned long long particlesProcessable = std::min(
			getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed + maxVectorsAllocatable),
			(unsigned long long)particleCount
		) - (vectorsProcessed > 0 ? getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed) : 0);
		if(particlesProcessable == 0) { 
			std::cout << "GPU can not run these many particles in Gravity\n";
			throw new std::runtime_error("GPU can not run these many particles in Gravity");
		}
		unsigned long long vectorsProcessable = getRowsAndColsCountMinusIdentityFromRows(particlesProcessed + particlesProcessable) - getRowsAndColsCountMinusIdentityFromRows(particlesProcessed);
		unsigned long long vectorsProcessableTriangular = vectorsProcessable / 2;

		radiusComponentKernel <<<1 + vectorsProcessableTriangular/256, 256>>> (td_par, accelerations, vectorsProcessableTriangular, G, vectorsProcessed / 2);
		cudaWithError->peekAtLastError("radiusComponentKernel");

		for(int i = particlesProcessed; i < particlesProcessable + particlesProcessed; i++) {
			addAccelerationsKernelLower <<<1 + i/256, 256>>> (td_par, accelerations, 0, i, i,  vectorsProcessed / 2);
			cudaWithError->peekAtLastError("addAccelerationsKernelLower");
			addAccelerationsKernelUpper <<<1 + (particleCount - 1 - i) / 256, 256>>> (
				td_par, accelerations, i + 1, i, particlesProcessed + particlesProcessable,  vectorsProcessed / 2, particlesProcessed, vectorsProcessableTriangular
			);
		}

		particlesProcessed += particlesProcessable;
		vectorsProcessed += vectorsProcessable;
	}
	
	cudaWithError->free(accelerations);
}

