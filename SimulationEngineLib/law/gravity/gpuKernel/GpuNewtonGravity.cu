#include "hip/hip_runtime.h"
#include "law/gravity/gpuKernel/GpuNewtonGravity.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "law/gravity/helper/GpuNewtonGravityHelper.cuh"

#include <cmath>
#include <algorithm>

GpuNewtonGravity::GpuNewtonGravity(float G) : GpuLaw("GpuNewtonGravity"), G(G) { }

__global__ 
void radiusComponentKernel(Particle** particles, Vector3D<float>* accelerations, unsigned long long betweenParticlesTriangularCount, float G, unsigned long long vectorsProcessedTriangular) {
	unsigned long long betweenParticlesTriangularIndex = threadIdx.x + blockIdx.x*blockDim.x;
	if(betweenParticlesTriangularIndex < betweenParticlesTriangularCount) { 
		radiusComponentKernelHelper(betweenParticlesTriangularIndex, particles, accelerations, betweenParticlesTriangularCount, G, vectorsProcessedTriangular);
	} 
}

__global__ 
void addAccelerationsKernelLower(Particle** particles, Vector3D<float>* accelerations, unsigned long long particleIndex2, unsigned long long vectorsProcessedTriangular) {
	unsigned long long particleIndex1 = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelLowerHelper(particleIndex1, particles, accelerations, particleIndex2, vectorsProcessedTriangular);
}

__global__ 
void addAccelerationsKernelUpper(Particle** particles, Vector3D<float>* accelerations, unsigned long long xOffset, unsigned long long particleIndex2, unsigned long long particleCount, unsigned long long vectorsProcessedTriangular, unsigned long long betweenParticlesTriangularCount) {
	unsigned long long particleIndex1 = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelUpperHelper(particleIndex1, particles, accelerations, xOffset, particleIndex2, particleCount, vectorsProcessedTriangular, betweenParticlesTriangularCount);
}

unsigned long long getRowsFromRowsAndColsCountMinusIdentity(unsigned long long rowsAndColsCount) {
	return 0.5 * (sqrt(4 * rowsAndColsCount + 1) + 1);
}

unsigned long long getRowsAndColsCountMinusIdentityFromRows(unsigned long long rowsAndColsCountMinusIdentityowsAndColsCount) {
	return (rowsAndColsCountMinusIdentityowsAndColsCount - 1) * rowsAndColsCountMinusIdentityowsAndColsCount;
}

void GpuNewtonGravity::run(Particle** particles, int particleCount) {
	unsigned long long betweenParticlesCount = ((unsigned long long)particleCount-1)*particleCount;
	Vector3D<float>* accelerations = NULL;
	
	long long freeGpuMemory = cudaWithError->getFreeGpuMemory();
	unsigned long long vector3DSize = sizeof(Vector3D<float>);
	long long maxVectorsAllocatableStage1 = freeGpuMemory / vector3DSize;
	long long maxVectorsAllocatable = std::min(maxVectorsAllocatableStage1, (long long)betweenParticlesCount);
	if(maxVectorsAllocatable <= 0) {
		throw std::runtime_error("Ran out of GPU memory");
	}

	cudaWithError->malloc((void**)&accelerations, maxVectorsAllocatable * sizeof(Vector3D<float>));

	unsigned long long particlesProcessed = 0;
	unsigned long long vectorsProcessed = 0;
	while(particlesProcessed < particleCount) {
		unsigned long long particlesProcessable = std::min(
			getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed + maxVectorsAllocatable),
			(unsigned long long)particleCount
		) - (vectorsProcessed > 0 ? getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed) : 0);
		if(particlesProcessable == 0) { 
			std::cout << "GPU can not run these many particles in Gravity\n";
			throw std::runtime_error("GPU can not run these many particles in Gravity");
		}
		unsigned long long vectorsProcessable = getRowsAndColsCountMinusIdentityFromRows(particlesProcessed + particlesProcessable) - getRowsAndColsCountMinusIdentityFromRows(particlesProcessed);
		unsigned long long vectorsProcessableTriangular = vectorsProcessable / 2;
		const unsigned int threadCount = 256;

		radiusComponentKernel <<<1 + vectorsProcessableTriangular/threadCount, threadCount>>> (particles, accelerations, vectorsProcessableTriangular, G, vectorsProcessed / 2);
		cudaWithError->peekAtLastError("radiusComponentKernel");

		for(int particleIndex = 0; particleIndex < particlesProcessed; particleIndex++) {
			addAccelerationsKernelUpper <<<1 + particlesProcessable / threadCount, threadCount>>> (
				particles, accelerations, std::max((int)particlesProcessed - particleIndex - 1, 0), particleIndex, particlesProcessed + particlesProcessable, vectorsProcessed / 2, vectorsProcessableTriangular
			);
			cudaWithError->peekAtLastError("addAccelerationsKernelUpper");
		}
		for(int particleIndex = particlesProcessed; particleIndex < particlesProcessable + particlesProcessed; particleIndex++) {
			addAccelerationsKernelLower <<<1 + particleIndex/threadCount, threadCount>>> (particles, accelerations, particleIndex, vectorsProcessed / 2);
			cudaWithError->peekAtLastError("addAccelerationsKernelLower");
			addAccelerationsKernelUpper <<<1 + particlesProcessable / threadCount, threadCount>>> (
				particles, accelerations, 0, particleIndex, particlesProcessed + particlesProcessable, vectorsProcessed / 2, vectorsProcessableTriangular
			);
			cudaWithError->peekAtLastError("addAccelerationsKernelUpper");
		}

		particlesProcessed += particlesProcessable;
		vectorsProcessed += vectorsProcessable;
	}
	
	cudaWithError->free(accelerations);
}

