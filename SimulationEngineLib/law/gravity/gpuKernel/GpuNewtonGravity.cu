#include "hip/hip_runtime.h"
#include "law/gravity/gpuKernel/GpuNewtonGravity.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "law/gravity/helper/GpuNewtonGravityHelper.cuh"

#include <cmath>
#include <algorithm>

GpuNewtonGravity::GpuNewtonGravity(float G) : GpuLaw("GpuNewtonGravity"), G(G) { }

#if defined(USE_GPU) 
	__global__
	void radiusComponentKernel(Particle** particles, Vector3D<float>* accelerations, unsigned long long betweenParticlesTriangularCount, float G, unsigned long long vectorsProcessedTriangular) {
		unsigned long long betweenParticlesTriangularIndex = threadIdx.x + blockIdx.x*blockDim.x;
		if (betweenParticlesTriangularIndex < betweenParticlesTriangularCount) { 
			radiusComponentKernelHelper(betweenParticlesTriangularIndex, particles, accelerations, betweenParticlesTriangularCount, G, vectorsProcessedTriangular);
		} 
	}
#endif 

#if defined(USE_GPU) 
	__global__
	void addAccelerationsKernelLower(Particle** particles, Vector3D<float>* accelerations, unsigned long long particleIndex2, unsigned long long vectorsProcessedTriangular) {
		unsigned long long particleIndex1 = threadIdx.x + blockIdx.x*blockDim.x;
		addAccelerationsKernelLowerHelper(particleIndex1, particles, accelerations, particleIndex2, vectorsProcessedTriangular);
	}
#endif 

#if defined(USE_GPU) 
	__global__
	void addAccelerationsKernelUpper(Particle** particles, Vector3D<float>* accelerations, unsigned long long xOffset, unsigned long long particleIndex2, unsigned long long particleCount, unsigned long long vectorsProcessedTriangular, unsigned long long betweenParticlesTriangularCount) {
		unsigned long long particleIndex1 = threadIdx.x + blockIdx.x*blockDim.x;
		addAccelerationsKernelUpperHelper(particleIndex1, particles, accelerations, xOffset, particleIndex2, particleCount, vectorsProcessedTriangular, betweenParticlesTriangularCount);
	}
#endif 

unsigned long long getRowsFromRowsAndColsCountMinusIdentity(unsigned long long rowsAndColsCount) {
	return 0.5 * (sqrt(4 * rowsAndColsCount + 1) + 1);
}

unsigned long long getRowsAndColsCountMinusIdentityFromRows(unsigned long long rowsAndColsCountMinusIdentityowsAndColsCount) {
	return (rowsAndColsCountMinusIdentityowsAndColsCount - 1) * rowsAndColsCountMinusIdentityowsAndColsCount;
}

void GpuNewtonGravity::run(Particle** particles, int particleCount) {
	#if defined(USE_GPU) 
		unsigned long long betweenParticlesCount = ((unsigned long long)particleCount-1)*particleCount;
		Vector3D<float>* accelerations = NULL;
		
		const double maxFractionMemoryAllocatable = 0.5;
		const long long freeGpuMemoryStage1 = cudaWithError->getFreeGpuMemory();
		const long long freeGpuMemory = (long long)(freeGpuMemoryStage1 * maxFractionMemoryAllocatable);
		const unsigned long long vector3DSize = sizeof(Vector3D<float>);
		const long long maxVectorsAllocatableStage1 = freeGpuMemory / vector3DSize;
		const long long maxVectorsAllocatable = std::min(maxVectorsAllocatableStage1, (long long)betweenParticlesCount);
		if (maxVectorsAllocatable <= 0) {
			throw std::runtime_error("Ran out of GPU memory");
		}

		cudaWithError->malloc((void**)&accelerations, maxVectorsAllocatable * sizeof(Vector3D<float>));

		unsigned long long particlesProcessed = 0;
		unsigned long long vectorsProcessed = 0;
		while(particlesProcessed < particleCount) {
			unsigned long long particlesProcessable = std::min(
				getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed + maxVectorsAllocatable),
				(unsigned long long)particleCount
			) - (vectorsProcessed > 0 ? getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed) : 0);
			if (particlesProcessable == 0) { 
				std::cout << "GPU can not run these many particles in Gravity\n";
				throw std::runtime_error("GPU can not run these many particles in Gravity");
			}
			unsigned long long vectorsProcessable = getRowsAndColsCountMinusIdentityFromRows(particlesProcessed + particlesProcessable) - getRowsAndColsCountMinusIdentityFromRows(particlesProcessed);
			unsigned long long vectorsProcessableTriangular = vectorsProcessable / 2;
			
			cudaWithError->runKernel("radiusComponentKernel", [&](unsigned int kernelSize) {
				radiusComponentKernel <<<1 + vectorsProcessableTriangular / kernelSize, kernelSize>>> (particles, accelerations, vectorsProcessableTriangular, G, vectorsProcessed / 2);
			});
			for(int particleIndex = 0; particleIndex < particlesProcessed; particleIndex++) {
				cudaWithError->runKernel("addAccelerationsKernelUpper", [&](unsigned int kernelSize) {
					addAccelerationsKernelUpper <<<1 + particlesProcessable / kernelSize, kernelSize>>> (
						particles, accelerations, std::max((int)particlesProcessed - particleIndex - 1, 0), particleIndex, particlesProcessed + particlesProcessable, vectorsProcessed / 2, vectorsProcessableTriangular
					);				
				});
			}
			for(int particleIndex = particlesProcessed; particleIndex < particlesProcessable + particlesProcessed; particleIndex++) {
				cudaWithError->runKernel("addAccelerationsKernelLower", [&](unsigned int kernelSize) {
					addAccelerationsKernelLower <<<1 + particleIndex / kernelSize, kernelSize>>> (particles, accelerations, particleIndex, vectorsProcessed / 2);				
				});
				cudaWithError->runKernel("addAccelerationsKernelUpper", [&](unsigned int kernelSize) {
					addAccelerationsKernelUpper <<<1 + particlesProcessable / kernelSize, kernelSize>>> (
						particles, accelerations, 0, particleIndex, particlesProcessed + particlesProcessable, vectorsProcessed / 2, vectorsProcessableTriangular
					);				
				});
			}

			particlesProcessed += particlesProcessable;
			vectorsProcessed += vectorsProcessable;
		}
		
		cudaWithError->free(accelerations);
	#endif 
}

