#include "hip/hip_runtime.h"
#pragma once
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"

#include <cmath>

__device__ __host__
Vector3D getAcceleration(double mass, Vector3D radiusComponent) {	
	return mass * radiusComponent;
}

__device__ __host__
void runOnParticle(Particle* p1, Vector3D acceleration) {
	p1->velocity = p1->velocity + acceleration;
}

__device__ __host__ 
Vector3D getRadiusComponent(Particle* p1, Particle* p2, double G) {
	Vector3D displacement = p1->position - p2->position;
	double displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared <= pow(p1->radius + p2->radius, 2)) {
		return {0, 0, 0};
	} else {
		Vector3D unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}