#include "hip/hip_runtime.h"
#pragma once
#include "law/gravity/helper/NewtonGravityHelper.cuh"
#include "particle/ParticleSimple.h"
#include "particle/Particle.cuh"
#include "util/MatrixMaths.cuh"

#include <cmath>

__device__ __host__
Vector3D<float> getAcceleration(float mass, Vector3D<float> radiusComponent) {	
	return mass * radiusComponent;
}

__device__ __host__
void runOnParticle(Particle* p1, Vector3D<float> acceleration) {
	p1->velocity = p1->velocity + acceleration;
}

__device__ __host__ 
Vector3D<float> getRadiusComponent(Particle* p1, Particle* p2, float G) {
	Vector3D<float> displacement = p1->position - p2->position;
	float displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared <= pow(p1->radius + p2->radius, 2)) {
		return {0, 0, 0};
	} else {
		Vector3D<float> unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}