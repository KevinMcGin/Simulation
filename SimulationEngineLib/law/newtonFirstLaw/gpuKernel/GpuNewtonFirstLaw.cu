#include "hip/hip_runtime.h"
#include "law/newtonFirstLaw/gpuKernel/GpuNewtonFirstLaw.cuh"
#include "particle/Particle.cuh"

GpuNewtonFirstLaw::GpuNewtonFirstLaw() : GpuLaw("NewtonFirstLaw") { }

#if defined(USE_GPU) 
	__global__
	static void advanceParticles(Particle** particles, int particleCount) {
		int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
		if(particleIndex < particleCount) { 
			particles[particleIndex]->advance();
		} 
	}
#endif 

void GpuNewtonFirstLaw::run(Particle** particles, int particleCount) {
	#if defined(USE_GPU) 
		advanceParticles <<<1 + particleCount/256, 256>>> (particles, particleCount);
		cudaWithError->peekAtLastError("advanceParticles");
	#endif 
}
