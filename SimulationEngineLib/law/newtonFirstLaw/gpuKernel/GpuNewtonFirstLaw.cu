#include "hip/hip_runtime.h"
#include "law/newtonFirstLaw/gpuKernel/GpuNewtonFirstLaw.cuh"
#include "particle/Particle.cuh"

GpuNewtonFirstLaw::GpuNewtonFirstLaw() : GpuLaw("NewtonFirstLaw") { }

__global__ 
static void advanceParticles(Particle** particles, int particleCount) {
	int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
	if(particleIndex < particleCount) { 
		particles[particleIndex]->advance();
	} 
}

void GpuNewtonFirstLaw::run(Particle** particles, int particleCount) {
	advanceParticles <<<1 + particleCount/256, 256>>> (particles, particleCount);
	cudaWithError->peekAtLastError("advanceParticles");
}
