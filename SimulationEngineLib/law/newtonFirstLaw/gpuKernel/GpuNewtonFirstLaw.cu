#include "hip/hip_runtime.h"
#include "law/newtonFirstLaw/gpuKernel/GpuNewtonFirstLaw.cuh"
#include "particle/Particle.cuh"

GpuNewtonFirstLaw::GpuNewtonFirstLaw() : GpuLaw("NewtonFirstLaw") { }

#if defined(USE_GPU) 
	__global__
	static void advanceParticles(Particle** particles, int particleCount) {
		int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
		if (particleIndex < particleCount) { 
			particles[particleIndex]->advance();
		} 
	}
#endif 

void GpuNewtonFirstLaw::run(Particle** particles, int particleCount) {
	#if defined(USE_GPU) 
		cudaWithError->runKernel("advanceParticles", [&](unsigned int kernelSize) {
			advanceParticles <<<1 + particleCount/kernelSize, kernelSize>>> (particles, particleCount);
		});
	#endif 
}
 