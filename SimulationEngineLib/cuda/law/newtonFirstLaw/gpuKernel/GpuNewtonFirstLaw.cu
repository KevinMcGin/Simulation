#include "hip/hip_runtime.h"
#include "cuda/law/newtonFirstLaw/gpuKernel/GpuNewtonFirstLaw.cuh"
#include "shared/particle/Particle.cuh"

GpuNewtonFirstLaw::GpuNewtonFirstLaw() : GpuLaw("NewtonFirstLaw") { }

__global__
static void advanceParticles(
	Particle** particles,
	int particleCount,
	unsigned int deltaTime
) {
	int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
	if (particleIndex < particleCount) { 
		particles[particleIndex]->advance(deltaTime);
	} 
}

void GpuNewtonFirstLaw::run(
	Particle** particles, 
	int particleCount,
	unsigned int deltaTime
) {
	cudaWithError->runKernel("advanceParticles", [&](unsigned int kernelSize) {
		advanceParticles <<<1 + particleCount/kernelSize, kernelSize>>> (
			particles,
			particleCount,
			deltaTime
		);
	});
}
 