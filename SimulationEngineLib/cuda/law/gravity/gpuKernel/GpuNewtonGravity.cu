#include "hip/hip_runtime.h"
#include "cuda/law/gravity/gpuKernel/GpuNewtonGravity.cuh"
#include "cpp/particle/ParticleSimple.h"
#include "shared/particle/Particle.cuh"
#include "shared/util/MatrixMaths.cuh"
#include "shared/law/gravity/helper/NewtonGravityHelper.cuh"
#include "cuda/law/gravity/helper/GpuNewtonGravityHelper.cuh"

#include <cmath>
#include <algorithm>

GpuNewtonGravity::GpuNewtonGravity(float G) : GpuLaw("GpuNewtonGravity"), G(G) { }

__global__
void radiusComponentKernel(Particle** particles, Vector3D<float>* accelerations, unsigned long long betweenParticlesTriangularCount, float G, unsigned long long vectorsProcessedTriangular) {
	unsigned long long betweenParticlesTriangularIndex = threadIdx.x + blockIdx.x*blockDim.x;
	if (betweenParticlesTriangularIndex < betweenParticlesTriangularCount) { 
		radiusComponentKernelHelper(betweenParticlesTriangularIndex, particles, accelerations, betweenParticlesTriangularCount, G, vectorsProcessedTriangular);
	} 
}

__global__
void addAccelerationsKernelLower(Particle** particles, Vector3D<float>* accelerations, unsigned long long particleIndex2, unsigned long long vectorsProcessedTriangular, unsigned int deltaTime) {
	unsigned long long particleIndex1 = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelLowerHelper(particleIndex1, particles, accelerations, particleIndex2, vectorsProcessedTriangular, deltaTime);
} 

__global__
void addAccelerationsKernelUpper(Particle** particles, Vector3D<float>* accelerations, unsigned long long xOffset, unsigned long long particleIndex2, unsigned long long particleCount, unsigned long long vectorsProcessedTriangular, unsigned long long betweenParticlesTriangularCount, unsigned int deltaTime) {
	unsigned long long particleIndex1 = threadIdx.x + blockIdx.x*blockDim.x;
	addAccelerationsKernelUpperHelper(particleIndex1, particles, accelerations, xOffset, particleIndex2, particleCount, vectorsProcessedTriangular, betweenParticlesTriangularCount, deltaTime);
}

unsigned long long getRowsFromRowsAndColsCountMinusIdentity(unsigned long long rowsAndColsCount) {
	return 0.5 * (sqrt(4 * rowsAndColsCount + 1) + 1);
}

unsigned long long getRowsAndColsCountMinusIdentityFromRows(unsigned long long rowsAndColsCountMinusIdentityowsAndColsCount) {
	return (rowsAndColsCountMinusIdentityowsAndColsCount - 1) * rowsAndColsCountMinusIdentityowsAndColsCount;
}

void GpuNewtonGravity::run(
	Particle** particles, 
	int particleCount,
	unsigned int deltaTime
) {
	unsigned long long betweenParticlesCount = ((unsigned long long)particleCount-1)*particleCount;
	Vector3D<float>* accelerations = NULL;
	
	const double maxFractionMemoryAllocatable = 0.95;
	const long long freeGpuMemoryStage1 = cudaWithError->getFreeGpuMemory();
	const long long freeGpuMemory = (long long)(freeGpuMemoryStage1 * maxFractionMemoryAllocatable);
	const unsigned long long vector3DSize = sizeof(Vector3D<float>);
	const long long maxVectorsAllocatableStage1 = freeGpuMemory / vector3DSize;
	const long long maxVectorsAllocatable = std::min(maxVectorsAllocatableStage1, (long long)betweenParticlesCount);
	if (maxVectorsAllocatable <= 0) {
		throw std::runtime_error("Ran out of GPU memory");
	}

	cudaWithError->malloc((void**)&accelerations, maxVectorsAllocatable * sizeof(Vector3D<float>));

	unsigned long long particlesProcessed = 0;
	unsigned long long vectorsProcessed = 0;
	while(particlesProcessed < particleCount) {
		unsigned long long particlesProcessable = std::min(
			getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed + maxVectorsAllocatable),
			(unsigned long long)particleCount
		) - (vectorsProcessed > 0 ? getRowsFromRowsAndColsCountMinusIdentity(vectorsProcessed) : 0);
		if (particlesProcessable == 0) { 
			std::cout << "GPU can not run these many particles in Gravity\n";
			throw std::runtime_error("GPU can not run these many particles in Gravity");
		}
		unsigned long long vectorsProcessable = getRowsAndColsCountMinusIdentityFromRows(particlesProcessed + particlesProcessable) - getRowsAndColsCountMinusIdentityFromRows(particlesProcessed);
		unsigned long long vectorsProcessableTriangular = vectorsProcessable / 2;
		
		cudaWithError->runKernel("radiusComponentKernel", [&](unsigned int kernelSize) {
			radiusComponentKernel <<<1 + vectorsProcessableTriangular / kernelSize, kernelSize>>> (particles, accelerations, vectorsProcessableTriangular, G, vectorsProcessed / 2);
		});
		for(int particleIndex = 0; particleIndex < particlesProcessed; particleIndex++) {
			cudaWithError->runKernel("addAccelerationsKernelUpper1", [&](unsigned int kernelSize) {
				addAccelerationsKernelUpper <<<1 + particlesProcessable / kernelSize, kernelSize>>> (
					particles, 
					accelerations, 
					std::max((int)particlesProcessed - particleIndex - 1, 0), 
					particleIndex, 
					particlesProcessed + particlesProcessable, 
					vectorsProcessed / 2, 
					vectorsProcessableTriangular,
					deltaTime
				);				
			});
		}
		for(int particleIndex = particlesProcessed; particleIndex < particlesProcessable + particlesProcessed; particleIndex++) {
			cudaWithError->runKernel("addAccelerationsKernelLower", [&](unsigned int kernelSize) {
				addAccelerationsKernelLower <<<1 + particleIndex / kernelSize, kernelSize>>> (
					particles, 
					accelerations, 
					particleIndex, 
					vectorsProcessed / 2,
					deltaTime
				);				
			});
			cudaWithError->runKernel("addAccelerationsKernelUpper2", [&](unsigned int kernelSize) {
				addAccelerationsKernelUpper <<<1 + particlesProcessable / kernelSize, kernelSize>>> (
					particles,
					accelerations, 
					0, 
					particleIndex, 
					particlesProcessed + particlesProcessable, 
					vectorsProcessed / 2, 
					vectorsProcessableTriangular,
					deltaTime
				);				
			});
		}

		particlesProcessed += particlesProcessable;
		vectorsProcessed += vectorsProcessable;
	}
	
	cudaWithError->free(accelerations);
}

