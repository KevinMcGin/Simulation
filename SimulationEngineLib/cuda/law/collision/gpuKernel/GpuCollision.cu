#include "hip/hip_runtime.h"
#include "cuda/law/collision/gpuKernel/GpuCollision.cuh"
#include "cuda/law/collision/helper/GpuCollisionHelper.cuh"
#include "cpp/particle/ParticleSimple.h"
#include "shared/law/collision/detector/CollisionDetectorSimple.cuh"
#include "shared/law/collision/resolver/CollisionResolverCoalesce.cuh"
#include "shared/util/MatrixMaths.cuh"

#include <assert.h>
#include <map>
#include <algorithm>
#include <iterator>
#include <cmath>
#include <typeinfo>

//Cuda doesn't recognise virtual functions of classes initialised on the CPU, so we have to initialise them here
#if defined(USE_GPU) 
	__global__
	void setCollisionDetector(CollisionDetector** collisionDetectorGpu, int collisionDetectorIndex) {
		int idx = threadIdx.x + blockIdx.x*blockDim.x;
		if (idx < 1) {
			if (collisionDetectorIndex == CollisionDetectorSimple::INDEX) {
				collisionDetectorGpu[0] = new CollisionDetectorSimple();
			} else {
				printf("collisionDetectorGpu could not be initialised\n");
				assert(false);
			}
		} 
	}
#endif 
#if defined(USE_GPU) 
	__global__
	void setCollisionResolver(CollisionResolver** collisionResolverGpu, int collisionResolverIndex) {
		int idx = threadIdx.x + blockIdx.x*blockDim.x;
		if (idx < 1) {
			if (collisionResolverIndex == CollisionResolverCoalesce::INDEX) {
				collisionResolverGpu[0] = new CollisionResolverCoalesce();
			} else {
				printf("collisionResolverGpu could not be initialised\n");
				assert(false);
			}
		} 
	} 
#endif 

GpuCollision::GpuCollision(std::shared_ptr<CollisionDetector> collisionDetector, std::shared_ptr<CollisionResolver> collisionResolver) : 
	GpuLaw("Collision") {
	cudaWithError->malloc((void**)&collisionDetectorGpu, sizeof(*collisionDetector));
	cudaWithError->malloc((void**)&collisionResolverGpu, sizeof(*collisionResolver));
	#if defined(USE_GPU) 
		cudaWithError->runKernel("setCollisionDetector", [&](unsigned int kernelSize) {
			setCollisionDetector <<<1, 1>>> (collisionDetectorGpu, collisionDetector->getIndex());
			setCollisionResolver <<<1, 1>>> (collisionResolverGpu, collisionResolver->getIndex());
		});
	#endif 
}

GpuCollision::~GpuCollision() {
	cudaWithError->free(collisionDetectorGpu);
	cudaWithError->free(collisionResolverGpu);
}

#if defined(USE_GPU) 
	__global__
	void getCollidedParticles(
		Particle** particles, 
		unsigned long long betweenParticlesOffset, 
		int* collisionMarks, 
		unsigned long long* collisionMarksIndex, 
		unsigned long long maxIntsAllocatable, 
		bool* particlesCollided, 
		CollisionDetector** collisionDetectorGpu, 
		unsigned long long n,
		const long long maxCollisionMarksIndex
	) {
		unsigned long long idx = threadIdx.x + (unsigned long long)blockIdx.x*blockDim.x;
		if (idx < n) { 
			getCollidedParticlesHelper(
				idx + betweenParticlesOffset, 
				particles, 
				collisionMarks, 
				collisionMarksIndex, 
				maxIntsAllocatable, 
				particlesCollided, 
				collisionDetectorGpu,
				maxCollisionMarksIndex
			);
		} 
	}
#endif 

#if defined(USE_GPU) 
	__global__
	void resolveCollidedParticles(
		Particle** particles, 
		int particlesOffset, 
		int* collisionMarks,  
		unsigned long long* collisionMarksIndex, 
		unsigned long long maxIntsAllocatable, 
		bool* particlesCollided, 
		CollisionResolver** collisionResolverGpu,
		int thisParticleCount, 
		int particleCount,		
		unsigned long long betweenParticlesOffset,
		unsigned long long thisBetweenParticleCount,
		const long long maxCollisionMarksIndex,
		bool* limitedReached
	) {
		int idx = threadIdx.x + blockIdx.x*blockDim.x;
		if (idx < thisParticleCount) { 
			resolveCollidedParticlesHelper(
				idx + particlesOffset,
				particles, 
				collisionMarks, 
				collisionMarksIndex, 
				maxIntsAllocatable, 
				particlesCollided, 
				collisionResolverGpu, 
				particleCount,
				betweenParticlesOffset,
				thisBetweenParticleCount,
				maxCollisionMarksIndex,
				limitedReached
			);
		} 
	}
#endif 

void GpuCollision::run(Particle** particles, int particleCount) {
	#if defined(USE_GPU) 
		unsigned long long betweenParticlesPairsCount = ((unsigned long long)particleCount-1)*particleCount;
		unsigned long long betweenParticlesCount = betweenParticlesPairsCount/2;

		unsigned long long* collisionMarksIndex = NULL;
		cudaWithError->malloc((void**)&collisionMarksIndex, sizeof(unsigned long long));

		bool* limitReached = NULL;
		cudaWithError->malloc((void**)&limitReached, sizeof(bool));
		
		bool* particlesCollided = NULL;
		cudaWithError->malloc((void**)&particlesCollided, particleCount * sizeof(bool));

		const double maxFractionMemoryAllocatable = 0.95;
		const long long freeGpuMemoryStage1 = cudaWithError->getFreeGpuMemory();
		const long long freeGpuMemory = (long long)(freeGpuMemoryStage1 * maxFractionMemoryAllocatable);
		const unsigned long long intSize = sizeof(int);
		const long long maxIntsAllocatableStage1 = freeGpuMemory / intSize;
		const unsigned long long maxIntsAllocatableFactor = 200;
		const long long maxIntsAllocatable = std::min(maxIntsAllocatableStage1, (long long)(betweenParticlesPairsCount * maxIntsAllocatableFactor));
		if (maxIntsAllocatable <= 0) {
			throw std::runtime_error("Ran out of GPU memory");
		}

		int* collisionMarks = NULL;
		cudaWithError->malloc((void**)&collisionMarks, maxIntsAllocatable * sizeof(int));

		const unsigned long long maxThreads = cudaWithError->getMaxThreads();
		const unsigned long long maxBetweenParticlesPerGetOverMaxThreads = 5;
		const unsigned long long maxBetweenParticlesPerGet = maxThreads * maxBetweenParticlesPerGetOverMaxThreads;
		const unsigned long long maxParticlesPerResolveMaxThreads = 5;
		const unsigned int maxParticlesPerResolve = (unsigned int)(maxThreads * maxParticlesPerResolveMaxThreads);
		unsigned long long collisionMarksIndexCpu = 0;
		unsigned long long maxCollisionMarksIndex = 2000;
		const unsigned long long minMaxCollisionMarksIndex = 5;
		const unsigned int maxLoops = 20000;
		unsigned int indexLoops = 0;
		bool limitReachedCpu = false;
		// std::cout << std::endl << "maxThreads: " << maxThreads << std::endl;
		do {
			limitReachedCpu = false;
			// std::cout << std::endl << "indexLoops: " << indexLoops << std::endl;
			if (++indexLoops > maxLoops) {
				std::cout << "Max Loops in GpuCollision reached" << std::endl;
				throw std::runtime_error("Max Loops in GpuCollision reached");
			}
			collisionMarksIndexCpu = 0;
			#if defined(USE_GPU)			
				cudaWithError->memcpy(collisionMarksIndex, &collisionMarksIndexCpu, sizeof(collisionMarksIndexCpu), hipMemcpyHostToDevice);
			#endif
			limitReachedCpu = false;
			cudaWithError->memcpy(limitReached, &limitReachedCpu, sizeof(limitReachedCpu), hipMemcpyHostToDevice);
			bool particlesCollidedDefault = false;
			#if defined(USE_GPU)
				for(int particleIndex = 0; particleIndex < particleCount; ++particleIndex) {
					cudaWithError->memcpy(&(particlesCollided[particleIndex]), &particlesCollidedDefault, sizeof(bool), hipMemcpyHostToDevice);
				}
			#endif
			for(unsigned long long betweenParticlesOffset = 0; betweenParticlesOffset < betweenParticlesCount; betweenParticlesOffset += maxBetweenParticlesPerGet) {
				const unsigned long long thisBetweenParticlesCount = std::min(maxBetweenParticlesPerGet, betweenParticlesCount - betweenParticlesOffset);
				#if defined(USE_GPU) 
					cudaWithError->runKernel("getCollidedParticles", [&](unsigned int kernelSize) {
						getCollidedParticles <<<1 + thisBetweenParticlesCount/kernelSize, kernelSize>>> (
							particles, 
							betweenParticlesOffset, 
							collisionMarks, 
							collisionMarksIndex,
							maxIntsAllocatable,
							particlesCollided, 
							collisionDetectorGpu, 
							thisBetweenParticlesCount,
							maxCollisionMarksIndex
						);
					});
				#endif
			}
			cudaWithError->deviceSynchronize("getCollidedParticles");
			for(int particlesOffset = 0; particlesOffset < particleCount; particlesOffset += maxParticlesPerResolve) {
				const int thisParticleCount = std::min(
					maxParticlesPerResolve, 
					(unsigned int)(particleCount - particlesOffset)
				);
				#if defined(USE_GPU)			
					cudaWithError->memcpy(&collisionMarksIndexCpu, collisionMarksIndex, sizeof(collisionMarksIndexCpu), hipMemcpyDeviceToHost);
				#endif
				collisionMarksIndexCpu = std::min(collisionMarksIndexCpu, maxCollisionMarksIndex);
				#if defined(USE_GPU)			
					cudaWithError->memcpy(collisionMarksIndex, &collisionMarksIndexCpu, sizeof(collisionMarksIndexCpu), hipMemcpyHostToDevice);
				#endif
				const unsigned int maxBetweenParticlesPerResolve = 10;
				for (unsigned long long betweenParticlesOffset = 0; betweenParticlesOffset < collisionMarksIndexCpu; betweenParticlesOffset += maxBetweenParticlesPerResolve) {
					const unsigned long long thisBetweenParticleCount = (unsigned long long)std::min(
						maxBetweenParticlesPerResolve, 
						(unsigned int)(collisionMarksIndexCpu - betweenParticlesOffset)
					);
					#if defined(USE_GPU)
						cudaWithError->runKernel("resolveCollidedParticles", [&](unsigned int kernelSize) {
							resolveCollidedParticles <<<1 + thisParticleCount/kernelSize, kernelSize>>> (
								particles, 
								particlesOffset,
								collisionMarks, 
								collisionMarksIndex, 
								maxIntsAllocatable,
								particlesCollided, 
								collisionResolverGpu, 
								thisParticleCount, 
								particleCount,
								betweenParticlesOffset,
								thisBetweenParticleCount,
								maxCollisionMarksIndex,
								limitReached
							);
						});
					#endif
				}
			}
			cudaWithError->deviceSynchronize("resolveCollidedParticles");
			// if max loop limited exceeded then reduce minMaxCollisionMarksIndex
			cudaWithError->memcpy(&limitReachedCpu, limitReached, sizeof(limitReachedCpu), hipMemcpyDeviceToHost);
			if (limitReachedCpu) {
				maxCollisionMarksIndex /= 2;
				// std::cout << std::endl << "maxCollisionMarksIndex reduced" << std::endl;
				if (maxCollisionMarksIndex < minMaxCollisionMarksIndex) {
					std::cout << std::endl << "maxCollisionMarksIndex < minMaxCollisionMarksIndex" << std::endl;
					// throw err;
				}
			}
			// std::cout << std::endl << "collisionMarksIndexCpu: " << collisionMarksIndexCpu << std::endl;
		} while (
			limitReachedCpu ||
			collisionMarksIndexCpu >= maxIntsAllocatable - 1 ||
			collisionMarksIndexCpu >= maxCollisionMarksIndex - 1
		);

		cudaWithError->free(collisionMarks);
		cudaWithError->free(collisionMarksIndex);
		cudaWithError->free(particlesCollided);
	#endif 
}