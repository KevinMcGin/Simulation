#include "hip/hip_runtime.h"
#include "cuda/gpuHelper/GpuDataController.cuh"
#include "cpp/particle/ParticleSimple.h"


GpuDataController::GpuDataController() : cudaWithError(CudaWithError("GpuDataController")) {
	cudaWithError.setDevice(0);
}

GpuDataController::~GpuDataController() {
	for(int i = 0; i < particleCount; i++) {
		cudaWithError.free(d_par[i]);
	}
	cudaWithError.free(td_par);
	delete d_par;
}

void GpuDataController::putParticlesOnDevice(std::vector<Particle*> particles, bool firstRun) {
	if (!firstRun) {
		for(int i = 0; i < particleCount; i++) {
			cudaWithError.free(d_par[i]);
		}
		cudaWithError.free(td_par);
		delete d_par;
	}
	
	particleCount = (int)particles.size();

	//Copy dynamically allocated child objects to GPU
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError.malloc((void**)&d_par[i], sizeof(*particles[i]));
		cudaWithError.memcpy(d_par[i], particles[i], sizeof(*particles[i]), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	cudaWithError.malloc((void**)&td_par, particleCount * sizeof(Particle*));
	cudaWithError.memcpy(td_par, d_par, particleCount * sizeof(Particle*), hipMemcpyHostToDevice);
}

void GpuDataController::getParticlesFromDevice(std::vector<Particle*>& particles) {
	cudaWithError.deviceSynchronize("getParticlesFromDevice");
	for(int i = 0; i < particleCount; i++) {
		cudaWithError.memcpy(particles[i], d_par[i], sizeof(*particles[i]), hipMemcpyDeviceToHost);
	}
	if (particleCount < particles.size()) {
		particles.erase (particles.begin() + particleCount, particles.end());
	}
}

__global__
static void deleteParticle(
	Particle** particles,
	int* particleCount
) {
	int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
	printf("particleCount: %d", *particleCount);
	if (particleIndex < *particleCount) { 
		if (particles[particleIndex]->deleted) {
			int lastParticleIndex = *particleCount - 1;
			printf("lastParticleIndex: %d", lastParticleIndex);
			if (particleIndex < lastParticleIndex) {
				// particles[particleIndex] = particles[lastParticleIndex];
			}
			// particles[lastParticleIndex] = nullptr;
			(*particleCount)--;
		}
	} 
}

void GpuDataController::deleteParticlesOnDevice() {
	int* gpuParticleCount = NULL;
	cudaWithError.malloc((void**)&gpuParticleCount, sizeof(particleCount));
	cudaWithError.memcpy(gpuParticleCount, &particleCount, sizeof(particleCount), hipMemcpyHostToDevice);

	int blockSize = 1;
	int numBlocks = particleCount;

	std::cout << "cpu: particleCount: " << particleCount;
	cudaWithError.runKernel("deleteParticlesOnDevice", [&](unsigned int kernelSize) {
		deleteParticle<<<numBlocks, blockSize>>>(td_par, gpuParticleCount);
	});
	
	cudaWithError.memcpy(&particleCount, gpuParticleCount, sizeof(particleCount), hipMemcpyDeviceToHost);
	
	cudaWithError.free(gpuParticleCount);
}

Particle** GpuDataController::get_td_par() { return td_par; }
int GpuDataController::getParticleCount() { return particleCount; }

