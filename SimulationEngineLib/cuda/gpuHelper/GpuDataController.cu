#include "hip/hip_runtime.h"
#include "cuda/gpuHelper/GpuDataController.cuh"
#include "cpp/particle/ParticleSimple.h"
#include "shared/particle/ParticleRelativistic.cuh"

GpuDataController::GpuDataController() : cudaWithError(CudaWithError("GpuDataController")) {
	cudaWithError.setDevice(0);
}

GpuDataController::~GpuDataController() {
	for(int i = 0; i < particleCount; i++) {
		cudaWithError.free(d_par[i]);
	}
	cudaWithError.free(td_par);
	delete d_par;
}

__global__
static void initParticle(
	Particle** particles,
	int particleCount
) {
	int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
	if (particleIndex < particleCount) { 
		particles[particleIndex] = new ParticleRelativistic(particles[particleIndex]);
	} 
}

void GpuDataController::putParticlesOnDevice(std::vector<Particle*> particles, bool firstRun) {
	if (!firstRun) {
		for(int i = 0; i < particleCount; i++) {
			cudaWithError.free(d_par[i]);
		}
		cudaWithError.free(td_par);
		delete d_par;
	}
	
	particleCount = (int)particles.size();

	//Copy dynamically allocated child objects to GPU
	d_par = new Particle*[particleCount];
	for(int i = 0; i < particleCount; ++i) {
		cudaWithError.malloc((void**)&d_par[i], sizeof(*particles[i]));
		cudaWithError.memcpy(d_par[i], particles[i], sizeof(*particles[i]), hipMemcpyHostToDevice);
	}

	//Copy the d_par array itself to the device
	cudaWithError.malloc((void**)&td_par, particleCount * sizeof(Particle*));
	cudaWithError.memcpy(td_par, d_par, particleCount * sizeof(Particle*), hipMemcpyHostToDevice);

	cudaWithError.runKernel("putParticlesOnDevice", [&](unsigned int kernelSize) {
		initParticle<<<1 + particleCount/kernelSize, kernelSize>>>(td_par, particleCount);
	});
}

void GpuDataController::getParticlesFromDevice(std::vector<Particle*>& particles) {
	cudaWithError.deviceSynchronize("getParticlesFromDevice");
	for(int i = 0; i < particleCount; i++) {
		cudaWithError.memcpy(particles[i], d_par[i], sizeof(*particles[i]), hipMemcpyDeviceToHost);
	}
	if (particleCount < particles.size()) {
		particles.erase (particles.begin() + particleCount, particles.end());
	}
}

__global__
static void deleteParticle(
	Particle** particles,
	int* particleCount
) {
	int particleIndex = threadIdx.x + blockIdx.x*blockDim.x;
	if (particleIndex < *particleCount) { 
		if (particles[particleIndex]->deleted) {
			int lastParticleIndex = *particleCount - 1;
			printf("lastParticleIndex: %d", lastParticleIndex);
			if (particleIndex < lastParticleIndex) {
				// particles[particleIndex] = particles[lastParticleIndex];
			}
			// particles[lastParticleIndex] = nullptr;
			(*particleCount)--;
		}
	} 
}

void GpuDataController::deleteParticlesOnDevice() {
	int* gpuParticleCount = NULL;
	cudaWithError.malloc((void**)&gpuParticleCount, sizeof(particleCount));
	cudaWithError.memcpy(gpuParticleCount, &particleCount, sizeof(particleCount), hipMemcpyHostToDevice);

	int blockSize = 1;
	int numBlocks = particleCount;

	cudaWithError.runKernel("deleteParticlesOnDevice", [&](unsigned int kernelSize) {
		deleteParticle<<<numBlocks, blockSize>>>(td_par, gpuParticleCount);
	});
	
	cudaWithError.memcpy(&particleCount, gpuParticleCount, sizeof(particleCount), hipMemcpyDeviceToHost);
	
	cudaWithError.free(gpuParticleCount);
}

Particle** GpuDataController::get_td_par() { return td_par; }
int GpuDataController::getParticleCount() { return particleCount; }

