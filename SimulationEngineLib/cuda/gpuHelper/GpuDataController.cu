#include "cuda/gpuHelper/GpuDataController.cuh"
#include "cpp/particle/ParticleSimple.h"

#if defined(USE_GPU)
	GpuDataController::GpuDataController() : cudaWithError(CudaWithError("GpuDataController")) {
		cudaWithError.setDevice(0);
	}

	GpuDataController::~GpuDataController() {
		for(int i = 0; i < particleCount; i++) {
			cudaWithError.free(d_par[i]);
		}
		cudaWithError.free(td_par);
		delete d_par;
	}

	void GpuDataController::putParticlesOnDevice(std::vector<Particle*> particles, bool firstRun) {
		if (!firstRun) {
			for(int i = 0; i < particleCount; i++) {
				cudaWithError.free(d_par[i]);
			}
			cudaWithError.free(td_par);
			delete d_par;
		}
		
		particleCount = (int)particles.size();

		//Copy dynamically allocated child objects to GPU
		d_par = new Particle*[particleCount];
		for(int i = 0; i < particleCount; ++i) {
			cudaWithError.malloc((void**)&d_par[i], sizeof(*particles[i]));
			cudaWithError.memcpy(d_par[i], particles[i], sizeof(*particles[i]), hipMemcpyHostToDevice);
		}

		//Copy the d_par array itself to the device
		cudaWithError.malloc((void**)&td_par, particleCount * sizeof(Particle*));
		cudaWithError.memcpy(td_par, d_par, particleCount * sizeof(Particle*), hipMemcpyHostToDevice);
	}

	void GpuDataController::getParticlesFromDevice(std::vector<Particle*>& particles) {
		cudaWithError.deviceSynchronize();
		for(int i = 0; i < particleCount; i++) {
			cudaWithError.memcpy(particles[i], d_par[i], sizeof(*particles[i]), hipMemcpyDeviceToHost);
		}
	}


	Particle** GpuDataController::get_td_par() { return td_par; }
	int GpuDataController::getParticleCount() { return particleCount; }

#else
	GpuDataController::GpuDataController() : cudaWithError(CudaWithError("GpuDataController")) {};
	GpuDataController::~GpuDataController() = default;

	void GpuDataController::putParticlesOnDevice(std::vector<Particle*> particles, bool firstRun) {}

	void GpuDataController::getParticlesFromDevice(std::vector<Particle*>& particles) {}

	Particle** GpuDataController::get_td_par() { return td_par; }
	int GpuDataController::getParticleCount() { return particleCount; }
#endif

