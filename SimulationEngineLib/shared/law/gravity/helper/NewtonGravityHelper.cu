#include "hip/hip_runtime.h"
#pragma once
#include "shared/law/gravity/helper/NewtonGravityHelper.cuh"
#include "cpp/particle/ParticleSimple.h"
#include "shared/particle/Particle.cuh"
#include "shared/util/MatrixMaths.cuh"

#include <cmath>

#if defined(USE_GPU)
   __device__ __host__
#endif
Vector3D<float> getAcceleration(float mass, Vector3D<float> radiusComponent) {	
	return mass * radiusComponent;
}

#if defined(USE_GPU)
   __device__ __host__
#endif
void runOnParticle(Particle* p1, Vector3D<float> acceleration, unsigned int deltaTime) {
	p1->velocity = p1->velocity + acceleration * deltaTime;
}

#if defined(USE_GPU)
   __device__ __host__
#endif 
Vector3D<float> getRadiusComponent(Particle* p1, Particle* p2, float G) {
	Vector3D<float> displacement = p1->position - p2->position;
	float displacementSquared = displacement.magnitudeSquared();
	if (displacementSquared <= pow(p1->radius + p2->radius, 2)) {
		return {0, 0, 0};
	} else {
		Vector3D<float> unit = displacement / sqrt(displacementSquared);
		return (G / displacementSquared) * unit;
	}
}