#include "hip/hip_runtime.h"
#include "shared/law/collision/resolver/CollisionResolverCoalesce.cuh"

#if defined(USE_GPU)
   __device__ __host__
#endif
void CollisionResolverCoalesce::resolve(Particle* p1, Particle* p2) {
	p1->radius = pow(pow(p1->radius, 3) + pow(p2->radius, 3), 1 / 3.0);
	p1->position = getCoalesced(p1->mass, p2->mass, p1->position, p2->position);
	p1->velocity = getCoalesced(p1->mass, p2->mass, p1->velocity, p2->velocity);
	// p1->angularVelocity = getCoalesced(p1->mass, p2->mass, p1->angularVelocity, p2->angularVelocity);
	p1->mass = p1->mass + p2->mass;
	p2->deleted = true;
}

#if defined(USE_GPU)
   __device__ __host__
#endif
Vector3D<float> CollisionResolverCoalesce::getCoalesced(float mass1, float mass2, Vector3D<float> vec1, Vector3D<float> vec2) {
	return (mass1 * vec1 + mass2 * vec2) / (mass1 + mass2);
}
