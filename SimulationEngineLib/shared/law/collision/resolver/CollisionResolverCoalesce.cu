#include "hip/hip_runtime.h"
#include "shared/law/collision/resolver/CollisionResolverCoalesce.cuh"

#if defined(USE_GPU)
	__device__ __host__
	#endif
	CollisionResolverCoalesce::CollisionResolverCoalesce(
		std::shared_ptr<MomentumService> momentumService
	) : CollisionResolver(momentumService) {}

#if defined(USE_GPU)
   __device__ __host__
#endif
void CollisionResolverCoalesce::resolve(
	Particle* p1, 
	Particle* p2,
	MomentumService* momentumService
) {
	p1->radius = pow(pow(p1->radius, 3) + pow(p2->radius, 3), 1 / 3.0);
	p1->position = getCoalesced(p1->mass, p2->mass, p1->position, p2->position);
	p1->velocity = getCoalescedVelocity(p1, p2);
	// p1->angularVelocity = getCoalesced(p1->mass, p2->mass, p1->angularVelocity, p2->angularVelocity);
	p1->mass = p1->mass + p2->mass;
	p2->deleted = true;
	p2->radius = 0;
	p2->mass = 0;
}

#if defined(USE_GPU)
   __device__ __host__
#endif
Vector3D<float> CollisionResolverCoalesce::getCoalesced(
	float mass1, 
	float mass2, 
	Vector3D<float> vec1, 
	Vector3D<float> vec2
) {
	return (mass1 * vec1 + mass2 * vec2) / (mass1 + mass2);
}

#if defined(USE_GPU)
   __device__ __host__
#endif
Vector3D<float> CollisionResolverCoalesce::getCoalescedVelocity(
	Particle* p1, 
	Particle* p2
) {
	return momentumService->mergeVelocity(
		p1->mass, 
		p1->velocity,
		p2->mass, 
		p2->velocity
	);
}


