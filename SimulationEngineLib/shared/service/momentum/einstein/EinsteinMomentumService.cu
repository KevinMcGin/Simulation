#include "hip/hip_runtime.h"
#include "shared/service/momentum/einstein/EinsteinMomentumService.cuh"

#if defined(USE_GPU)
__device__ __host__
#endif
float EinsteinMomentumService::getGamma(
    Vector3D<float> velocity
) {
    return 1 / 
        sqrt(
            1 - (velocity.magnitudeSquared() / speedLightSquared)
        );
}

#if defined(USE_GPU)
__device__ __host__
#endif
Vector3D<float> EinsteinMomentumService::getVelocityFromMomentum(
    float mass,
    Vector3D<float> momentum
) {
    return momentum / (
        sqrt(
            pow(mass, 2) + (momentum.magnitudeSquared() / speedLightSquared)
        )
    );
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getVelocityPlusAcceleration(
    float mass,
    Vector3D<float> acceleration, 
    unsigned int deltaTime,
    Vector3D<float> velocity
) {
    auto classicalVelocityChange = acceleration * deltaTime;
    auto classicalVelocityChangeMagnitudeSquared = classicalVelocityChange.magnitudeSquared();
    auto massMomentumChange = mass;
    // Condition for completeness sake. 
    // Condition rarely met. But if met, would otherwise create imaginary numbers in the gamma calculation.
    if (classicalVelocityChangeMagnitudeSquared > speedLightSquared) {
        //large offset is due to float errors
        float belowSpeedOfLight = (float)speedLight - 1000000.0f;
        classicalVelocityChange = belowSpeedOfLight * classicalVelocityChange.unit();
        massMomentumChange *= classicalVelocityChangeMagnitudeSquared / belowSpeedOfLight;
    }
    // Assumption: a small change in classical momentum is equal to a small change in relativistic
    auto relativisticMomentumChange = getMomentum(massMomentumChange, classicalVelocityChange);
    auto currentMomentum = getMomentum(mass, velocity);
    auto newMomentum = currentMomentum + relativisticMomentumChange;
    
    return getVelocityFromMomentum(mass, newMomentum);
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::mergeVelocity(
    float mass1, 
    Vector3D<float> velocity1,
    float mass2, 
    Vector3D<float> velocity2
) {
    auto p =  (
        getMomentum(mass1, velocity1) + 
        getMomentum(mass2, velocity2)
    );
    return getVelocityFromMomentum(mass1 + mass2, p);
}


#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getMomentum(
    float mass, 
    Vector3D<float> velocity
) {
    return getGamma(velocity) * NewtonMomentumService::getMomentum(mass, velocity);
}
