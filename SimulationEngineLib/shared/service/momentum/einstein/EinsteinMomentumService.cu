#include "hip/hip_runtime.h"
#include "shared/service/momentum/einstein/EinsteinMomentumService.cuh"

#if defined(USE_GPU)
__device__ __host__
#endif
float EinsteinMomentumService::getGamma(
    Vector3D<float> velocity
) {
    return 1 / 
        sqrt(
            1 - (velocity.magnitudeSquared() / speedLightSquared)
        );
}

#if defined(USE_GPU)
__device__ __host__
#endif
Vector3D<float> EinsteinMomentumService::getVelocityFromMomentum(
    float mass,
    Vector3D<float> momentum
) {
    return momentum / (
        sqrt(
            pow(mass, 2) + (momentum.magnitudeSquared() / speedLightSquared)
        )
    );
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getVelocityPlusAcceleration(
    float mass,
    Vector3D<float> acceleration, 
    unsigned int deltaTime,
    Vector3D<float> velocity
) {
    auto classicalVelocityChange = acceleration * deltaTime;
    auto classicalVelocityChangeMagnitudeSquared = classicalVelocityChange.magnitudeSquared();
    if (classicalVelocityChangeMagnitudeSquared > speedLightSquared) {
        //offset is due to float errors
        float belowSpeedOfLight = (float)speedLight - 1000000.0f;
        classicalVelocityChange = belowSpeedOfLight * classicalVelocityChange.unit();
        mass *= classicalVelocityChangeMagnitudeSquared / belowSpeedOfLight;
    }
    auto relativisticMomentumChange = getMomentum(mass, classicalVelocityChange);
    auto currentMomentum = getMomentum(mass, velocity);
    auto newMomentum = currentMomentum + relativisticMomentumChange;
    
    return getVelocityFromMomentum(mass, newMomentum);
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::mergeVelocity(
    float mass1, 
    Vector3D<float> velocity1,
    float mass2, 
    Vector3D<float> velocity2
) {
    auto p =  (
        getMomentum(mass1, velocity1) + 
        getMomentum(mass2, velocity2)
    );
    return getVelocityFromMomentum(mass1 + mass2, p);
}


#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getMomentum(
    float mass, 
    Vector3D<float> velocity
) {
    return getGamma(velocity) * 
        NewtonMomentumService::getMomentum(mass, velocity);
}
