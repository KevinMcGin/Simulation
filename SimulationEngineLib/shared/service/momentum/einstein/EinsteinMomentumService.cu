#include "hip/hip_runtime.h"
#include "shared/service/momentum/einstein/EinsteinMomentumService.cuh"

#if defined(USE_GPU)
__device__ __host__
#endif
float EinsteinMomentumService::getGamma(
    Vector3D<float> velocity
) {
    return 1 / sqrt(1 - velocity.magnitudeSquared() / speedLightSquared);
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::addMomentum(
    Vector3D<float> acceleration, 
    unsigned int deltaTime,
    float mass,
    Vector3D<float> velocity
) {
    auto classicalForce = acceleration * mass;
    auto realtivisticForce = pow(getGamma(velocity), 3) * classicalForce;
    return NewtonMomentumService::addMomentum(
        realtivisticForce / mass, 
        deltaTime,
        mass,
        velocity
    );
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::mergeVelocity(
    float mass1, 
    Vector3D<float> velocity1,
    float mass2, 
    Vector3D<float> velocity2
) {
    return (
        getMomentum(mass1, velocity1) + 
        getMomentum(mass2, velocity2)
    ) / (mass1 + mass2);

    auto u = NewtonMomentumService::getMomentum(mass1, velocity1) + 
        NewtonMomentumService::getMomentum(mass2, velocity2);
    return speedLight * u / sqrt(speedLightSquared + u.magnitudeSquared());
}


#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getMomentum(
    float mass, 
    Vector3D<float> velocity
) {
    return getGamma(velocity) * 
        NewtonMomentumService::getMomentum(mass, velocity);
}
