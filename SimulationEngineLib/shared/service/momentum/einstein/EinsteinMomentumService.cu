#include "hip/hip_runtime.h"
#include "shared/service/momentum/einstein/EinsteinMomentumService.cuh"

#if defined(USE_GPU)
__device__ __host__
#endif
float EinsteinMomentumService::getGamma(
    Vector3D<float> velocity
) {
    return 1 / 
        sqrt(
            1 - (velocity.magnitudeSquared() / speedLightSquared)
        );
}

#if defined(USE_GPU)
__device__ __host__
#endif
Vector3D<float> EinsteinMomentumService::getVelocityFromMomentum(
    float mass,
    Vector3D<float> momentum
) {
    return momentum / (
        sqrt(
            pow(mass, 2) + (momentum.magnitudeSquared() / speedLightSquared)
        )
    );
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getVelocityPlusAcceleration(
    float mass,
    Vector3D<float> acceleration, 
    unsigned int deltaTime,
    Vector3D<float> velocity
) {
    auto classicalVelocityChange = acceleration * deltaTime;
    auto relativisticVelocityChange = classicalVelocityChange / pow(getGamma(velocity), 3);
    
    return velocity + relativisticVelocityChange;
}

#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::mergeVelocity(
    float mass1, 
    Vector3D<float> velocity1,
    float mass2, 
    Vector3D<float> velocity2
) {
    auto p =  (
        getMomentum(mass1, velocity1) + 
        getMomentum(mass2, velocity2)
    );
    return getVelocityFromMomentum(mass1 + mass2, p);
}


#if defined(USE_GPU)
__device__ __host__
#endif 
Vector3D<float> EinsteinMomentumService::getMomentum(
    float mass, 
    Vector3D<float> velocity
) {
    return getGamma(velocity) * 
        NewtonMomentumService::getMomentum(mass, velocity);
}
