#include "hip/hip_runtime.h"
#include "shared/particle/ParticleRelativistic.cuh"


#if defined(USE_GPU)
    __device__ __host__
#endif
ParticleRelativistic::ParticleRelativistic(Particle* p) : Particle(p) {}

#if defined(USE_GPU)
    __device__ __host__
#endif
ParticleRelativistic::ParticleRelativistic(
    float			  mass,
    float            radius,
    Vector3D<float>  position,
    Vector3D<float>  velocity,
    bool             deleted
) : Particle(mass, radius, position, velocity, deleted) {}

#if defined(USE_GPU)
__device__ __host__
#endif
float ParticleRelativistic::getGamma() {
    return 1 / sqrt(1 - velocity.magnitudeSquared() / speedLightSquared);
}

#if defined(USE_GPU)
__device__ __host__
#endif
void ParticleRelativistic::addMomentum(
    Vector3D<float> acceleration, 
    unsigned int deltaTime
) {
    auto classicalForce = acceleration * mass;
    auto realtivisticForce = pow(getGamma(), 3) * classicalForce;
    Particle::addMomentum(realtivisticForce / mass, deltaTime);
}

#if defined(USE_GPU)
__device__ __host__
#endif
Vector3D<float> ParticleRelativistic::getMomentum() {
	return getGamma() * mass * velocity;
}

#if defined(USE_GPU)
__device__ __host__
#endif
Vector3D<float> ParticleRelativistic::mergeVelocity(Particle* p) {
	auto u = (getMomentum() + p->getMomentum()) / 
		(mass + p->mass);
    return speedLight * u / sqrt(speedLightSquared + u.magnitudeSquared());
}
